#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <functional>
#include <vector>
#include <iostream>
#include <stdint.h>
#ifndef _MSC_VER
#include <mm_malloc.h>
#endif
#include "sa_cuda_context.hpp"
#include "param.h"
#include "sa_blake.h"



#define THRD 64
#define BLK (NR_ROWS/THRD)


#define WN PARAM_N
#define WK PARAM_K

#define COLLISION_BIT_LENGTH (WN / (WK+1))
#define COLLISION_BYTE_LENGTH ((COLLISION_BIT_LENGTH+7)/8)
#define FINAL_FULL_WIDTH (2*COLLISION_BYTE_LENGTH+sizeof(uint32_t)*(1 << (WK)))

#define NDIGITS   (WK+1)
#define DIGITBITS (WN/(NDIGITS))
#define PROOFSIZE (1u<<WK)
#define COMPRESSED_PROOFSIZE ((COLLISION_BIT_LENGTH+1)*PROOFSIZE*4/(8*sizeof(uint32_t)))

typedef uint32_t uint;
typedef uint8_t uchar;
typedef uint64_t ulong;
typedef uint16_t ushort;
typedef uint32_t u32;

//orig defines

#if NR_ROWS_LOG <= 16 && NR_SLOTS <= (1 << 8)

#define ENCODE_INPUTS(row, slot0, slot1) \
    ((row << 16) | ((slot1 & 0xff) << 8) | (slot0 & 0xff))
#define DECODE_ROW(REF)   (REF >> 16)
#define DECODE_SLOT1(REF) ((REF >> 8) & 0xff)
#define DECODE_SLOT0(REF) (REF & 0xff)

#elif NR_ROWS_LOG == 18 && NR_SLOTS <= (1 << 7)

#define ENCODE_INPUTS(row, slot0, slot1) \
    ((row << 14) | ((slot1 & 0x7f) << 7) | (slot0 & 0x7f))
#define DECODE_ROW(REF)   (REF >> 14)
#define DECODE_SLOT1(REF) ((REF >> 7) & 0x7f)
#define DECODE_SLOT0(REF) (REF & 0x7f)

#elif NR_ROWS_LOG == 19 && NR_SLOTS <= (1 << 6)

#define ENCODE_INPUTS(row, slot0, slot1) \
    ((row << 13) | ((slot1 & 0x3f) << 6) | (slot0 & 0x3f)) /* 1 spare bit */
#define DECODE_ROW(REF)   (REF >> 13)
#define DECODE_SLOT1(REF) ((REF >> 6) & 0x3f)
#define DECODE_SLOT0(REF) (REF & 0x3f)

#elif NR_ROWS_LOG == 20 && NR_SLOTS <= (1 << 6)

#define ENCODE_INPUTS(row, slot0, slot1) \
    ((row << 12) | ((slot1 & 0x3f) << 6) | (slot0 & 0x3f))
#define DECODE_ROW(REF)   (REF >> 12)
#define DECODE_SLOT1(REF) ((REF >> 6) & 0x3f)
#define DECODE_SLOT0(REF) (REF & 0x3f)

#else
#error "unsupported NR_ROWS_LOG"
#endif




__constant__ ulong blake_iv[] =
{
	0x6a09e667f3bcc908, 0xbb67ae8584caa73b,
	0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1,
	0x510e527fade682d1, 0x9b05688c2b3e6c1f,
	0x1f83d9abfb41bd6b, 0x5be0cd19137e2179,
};

//OPENCL TO CUDA

#define __global
#define __local __shared__
#define get_global_id(F) (blockIdx.x * blockDim.x + threadIdx.x)
#define get_global_size(F) (gridDim.x * blockDim.x)
#define get_local_id(F) (threadIdx.x)
#define get_local_size(F) (blockDim.x)
#define barrier(F) __syncthreads()

//#define barrier(F) __threadfence()

#define atomic_add(A,X) atomicAdd(A,X)
#define atomic_inc(A) atomic_add(A,1)
#define atomic_sub(A,X) atomicSub(A,X)

__global__
void kernel_init_ht(__global char *ht, __global uint *rowCounters)
{
    rowCounters[get_global_id(0)] = 0;
}


__device__ uint ht_store(uint round, char *ht, uint i, ulong xi0, ulong xi1, ulong xi2, ulong xi3,uint *rowCounters)
{
	    uint    row;
    __global char       *p;
    uint                cnt;
#if NR_ROWS_LOG == 16
    if (!(round % 2))
	row = (xi0 & 0xffff);
    else
	// if we have in hex: "ab cd ef..." (little endian xi0) then this
	// formula computes the row as 0xdebc. it skips the 'a' nibble as it
	// is part of the PREFIX. The Xi will be stored starting with "ef...";
	// 'e' will be considered padding and 'f' is part of the current PREFIX
	row = ((xi0 & 0xf00) << 4) | ((xi0 & 0xf00000) >> 12) |
	    ((xi0 & 0xf) << 4) | ((xi0 & 0xf000) >> 12);
#elif NR_ROWS_LOG == 18
    if (!(round % 2))
	row = (xi0 & 0xffff) | ((xi0 & 0xc00000) >> 6);
    else
	row = ((xi0 & 0xc0000) >> 2) |
	    ((xi0 & 0xf00) << 4) | ((xi0 & 0xf00000) >> 12) |
	    ((xi0 & 0xf) << 4) | ((xi0 & 0xf000) >> 12);
#elif NR_ROWS_LOG == 19
    if (!(round % 2))
	row = (xi0 & 0xffff) | ((xi0 & 0xe00000) >> 5);
    else
	row = ((xi0 & 0xe0000) >> 1) |
	    ((xi0 & 0xf00) << 4) | ((xi0 & 0xf00000) >> 12) |
	    ((xi0 & 0xf) << 4) | ((xi0 & 0xf000) >> 12);
#elif NR_ROWS_LOG == 20
    if (!(round % 2))
	row = (xi0 & 0xffff) | ((xi0 & 0xf00000) >> 4);
    else
	row = ((xi0 & 0xf0000) >> 0) |
	    ((xi0 & 0xf00) << 4) | ((xi0 & 0xf00000) >> 12) |
	    ((xi0 & 0xf) << 4) | ((xi0 & 0xf000) >> 12);
#else
#error "unsupported NR_ROWS_LOG"
#endif
    xi0 = (xi0 >> 16) | (xi1 << (64 - 16));
    xi1 = (xi1 >> 16) | (xi2 << (64 - 16));
    xi2 = (xi2 >> 16) | (xi3 << (64 - 16));
    p = ht + row * NR_SLOTS * SLOT_LEN;
    uint rowIdx = row/ROWS_PER_UINT;
    uint rowOffset = BITS_PER_ROW*(row%ROWS_PER_UINT);
    uint xcnt = atomic_add(rowCounters + rowIdx, 1 << rowOffset);
    xcnt = (xcnt >> rowOffset) & ROW_MASK;
    cnt = xcnt;
     if (cnt >= NR_SLOTS)
       {
 	// avoid overflows
 	atomic_sub(rowCounters + rowIdx, 1 << rowOffset);
  	return 1;
       }

    p += cnt * SLOT_LEN + xi_offset_for_round(round);
    // store "i" (always 4 bytes before Xi)
    *(__global uint *)(p - 4) = i;
    if (round == 0 || round == 1)
      {
	// store 24 bytes
	*(__global ulong *)(p + 0) = xi0;
	*(__global ulong *)(p + 8) = xi1;
	*(__global ulong *)(p + 16) = xi2;
      }
    else if (round == 2)
      {
	// store 20 bytes
	*(__global uint *)(p + 0) = xi0;
	*(__global ulong *)(p + 4) = (xi0 >> 32) | (xi1 << 32);
	*(__global ulong *)(p + 12) = (xi1 >> 32) | (xi2 << 32);
      }
    else if (round == 3)
      {
	// store 16 bytes
	*(__global uint *)(p + 0) = xi0;
	*(__global ulong *)(p + 4) = (xi0 >> 32) | (xi1 << 32);
	*(__global uint *)(p + 12) = (xi1 >> 32);
      }
    else if (round == 4)
      {
	// store 16 bytes
	*(__global ulong *)(p + 0) = xi0;
	*(__global ulong *)(p + 8) = xi1;
      }
    else if (round == 5)
      {
	// store 12 bytes
	*(__global ulong *)(p + 0) = xi0;
	*(__global uint *)(p + 8) = xi1;
      }
    else if (round == 6 || round == 7)
      {
	// store 8 bytes
	*(__global uint *)(p + 0) = xi0;
	*(__global uint *)(p + 4) = (xi0 >> 32);
      }
    else if (round == 8)
      {
	// store 4 bytes
	*(__global uint *)(p + 0) = xi0;
      }
    return 0;
}


#define rotate(a, bits) ((a) << (bits)) | ((a) >> (64 - (bits)))

#define mix(va, vb, vc, vd, x, y) \
    va = (va + vb + x); \
    vd = rotate((vd ^ va), 64 - 32); \
    vc = (vc + vd); \
    vb = rotate((vb ^ vc), 64 - 24); \
    va = (va + vb + y); \
    vd = rotate((vd ^ va), 64 - 16); \
    vc = (vc + vd); \
    vb = rotate((vb ^ vc), 64 - 63);

__global__ 
void kernel_round0(__global ulong *blake_state, __global char *ht,
	__global uint *rowCounters, __global uint *debug)
{
    uint                tid = get_global_id(0);
    ulong               v[16];
    uint                inputs_per_thread = NR_INPUTS / get_global_size(0);
    uint                input = tid * inputs_per_thread;
    uint                input_end = (tid + 1) * inputs_per_thread;
    uint                dropped = 0;
    while (input < input_end)
      {
	// shift "i" to occupy the high 32 bits of the second ulong word in the
	// message block
	ulong word1 = (ulong)input << 32;
	// init vector v
	v[0] = blake_state[0];
	v[1] = blake_state[1];
	v[2] = blake_state[2];
	v[3] = blake_state[3];
	v[4] = blake_state[4];
	v[5] = blake_state[5];
	v[6] = blake_state[6];
	v[7] = blake_state[7];
	v[8] =  blake_iv[0];
	v[9] =  blake_iv[1];
	v[10] = blake_iv[2];
	v[11] = blake_iv[3];
	v[12] = blake_iv[4];
	v[13] = blake_iv[5];
	v[14] = blake_iv[6];
	v[15] = blake_iv[7];
	// mix in length of data
	v[12] ^= ZCASH_BLOCK_HEADER_LEN + 4 /* length of "i" */;
	// last block
	v[14] ^= (ulong)-1;

	// round 1
	mix(v[0], v[4], v[8],  v[12], 0, word1);
	mix(v[1], v[5], v[9],  v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8],  v[13], 0, 0);
	mix(v[3], v[4], v[9],  v[14], 0, 0);
	// round 2
	mix(v[0], v[4], v[8],  v[12], 0, 0);
	mix(v[1], v[5], v[9],  v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], word1, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8],  v[13], 0, 0);
	mix(v[3], v[4], v[9],  v[14], 0, 0);
	// round 3
	mix(v[0], v[4], v[8],  v[12], 0, 0);
	mix(v[1], v[5], v[9],  v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8],  v[13], 0, word1);
	mix(v[3], v[4], v[9],  v[14], 0, 0);
	// round 4
	mix(v[0], v[4], v[8],  v[12], 0, 0);
	mix(v[1], v[5], v[9],  v[13], 0, word1);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8],  v[13], 0, 0);
	mix(v[3], v[4], v[9],  v[14], 0, 0);
	// round 5
	mix(v[0], v[4], v[8],  v[12], 0, 0);
	mix(v[1], v[5], v[9],  v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, word1);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8],  v[13], 0, 0);
	mix(v[3], v[4], v[9],  v[14], 0, 0);
	// round 6
	mix(v[0], v[4], v[8],  v[12], 0, 0);
	mix(v[1], v[5], v[9],  v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8],  v[13], 0, 0);
	mix(v[3], v[4], v[9],  v[14], word1, 0);
	// round 7
	mix(v[0], v[4], v[8],  v[12], 0, 0);
	mix(v[1], v[5], v[9],  v[13], word1, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8],  v[13], 0, 0);
	mix(v[3], v[4], v[9],  v[14], 0, 0);
	// round 8
	mix(v[0], v[4], v[8],  v[12], 0, 0);
	mix(v[1], v[5], v[9],  v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, word1);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8],  v[13], 0, 0);
	mix(v[3], v[4], v[9],  v[14], 0, 0);
	// round 9
	mix(v[0], v[4], v[8],  v[12], 0, 0);
	mix(v[1], v[5], v[9],  v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8],  v[13], word1, 0);
	mix(v[3], v[4], v[9],  v[14], 0, 0);
	// round 10
	mix(v[0], v[4], v[8],  v[12], 0, 0);
	mix(v[1], v[5], v[9],  v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], word1, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8],  v[13], 0, 0);
	mix(v[3], v[4], v[9],  v[14], 0, 0);
	// round 11
	mix(v[0], v[4], v[8],  v[12], 0, word1);
	mix(v[1], v[5], v[9],  v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8],  v[13], 0, 0);
	mix(v[3], v[4], v[9],  v[14], 0, 0);
	// round 12
	mix(v[0], v[4], v[8],  v[12], 0, 0);
	mix(v[1], v[5], v[9],  v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], word1, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8],  v[13], 0, 0);
	mix(v[3], v[4], v[9],  v[14], 0, 0);

	// compress v into the blake state; this produces the 50-byte hash
	// (two Xi values)
	ulong h[7];
	h[0] = blake_state[0] ^ v[0] ^ v[8];
	h[1] = blake_state[1] ^ v[1] ^ v[9];
	h[2] = blake_state[2] ^ v[2] ^ v[10];
	h[3] = blake_state[3] ^ v[3] ^ v[11];
	h[4] = blake_state[4] ^ v[4] ^ v[12];
	h[5] = blake_state[5] ^ v[5] ^ v[13];
	h[6] = (blake_state[6] ^ v[6] ^ v[14]) & 0xffff;

	// store the two Xi values in the hash table
#if ZCASH_HASH_LEN == 50
	dropped += ht_store(0, ht, input * 2,
		h[0],
		h[1],
		h[2],
		h[3], rowCounters);
	dropped += ht_store(0, ht, input * 2 + 1,
		(h[3] >> 8) | (h[4] << (64 - 8)),
		(h[4] >> 8) | (h[5] << (64 - 8)),
		(h[5] >> 8) | (h[6] << (64 - 8)),
		(h[6] >> 8), rowCounters);
#else
#error "unsupported ZCASH_HASH_LEN"
#endif

	input++;
      }
#ifdef ENABLE_DEBUG
    debug[tid * 2] = 0;
    debug[tid * 2 + 1] = dropped;
#endif
}


__device__ ulong half_aligned_long(ulong *p, uint offset)
{
	return
		(((ulong)*(uint *)((char *)p + offset + 0)) << 0) |
		(((ulong)*(uint *)((char *)p + offset + 4)) << 32);
}

/*
** Access a well-aligned int.
*/
__device__ uint well_aligned_int(ulong *_p, uint offset)
{
	char *p = (char *)_p;
	return *(uint *)(p + offset);
}

/*
** XOR a pair of Xi values computed at "round - 1" and store the result in the
** hash table being built for "round". Note that when building the table for
** even rounds we need to skip 1 padding byte present in the "round - 1" table
** (the "0xAB" byte mentioned in the description at the top of this file.) But
** also note we can't load data directly past this byte because this would
** cause an unaligned memory access which is undefined per the OpenCL spec.
**
** Return 0 if successfully stored, or 1 if the row overflowed.
*/
__device__ 
uint xor_and_store(uint round, __global char *ht_dst, uint row,
	uint slot_a, uint slot_b, __global ulong *a, __global ulong *b,
	__global uint *rowCounters)
{
    ulong xi0, xi1, xi2;
#if NR_ROWS_LOG >= 16 && NR_ROWS_LOG <= 20
    // Note: for NR_ROWS_LOG == 20, for odd rounds, we could optimize by not
    // storing the byte containing bits from the previous PREFIX block for
    if (round == 1 || round == 2)
      {
	// xor 24 bytes
	xi0 = *(a++) ^ *(b++);
	xi1 = *(a++) ^ *(b++);
	xi2 = *a ^ *b;
	if (round == 2)
	  {
	    // skip padding byte
	    xi0 = (xi0 >> 8) | (xi1 << (64 - 8));
	    xi1 = (xi1 >> 8) | (xi2 << (64 - 8));
	    xi2 = (xi2 >> 8);
	  }
      }
    else if (round == 3)
      {
	// xor 20 bytes
	xi0 = half_aligned_long(a, 0) ^ half_aligned_long(b, 0);
	xi1 = half_aligned_long(a, 8) ^ half_aligned_long(b, 8);
	xi2 = well_aligned_int(a, 16) ^ well_aligned_int(b, 16);
      }
    else if (round == 4 || round == 5)
      {
	// xor 16 bytes
	xi0 = half_aligned_long(a, 0) ^ half_aligned_long(b, 0);
	xi1 = half_aligned_long(a, 8) ^ half_aligned_long(b, 8);
	xi2 = 0;
	if (round == 4)
	  {
	    // skip padding byte
	    xi0 = (xi0 >> 8) | (xi1 << (64 - 8));
	    xi1 = (xi1 >> 8);
	  }
      }
    else if (round == 6)
      {
	// xor 12 bytes
	xi0 = *a++ ^ *b++;
	xi1 = *(__global uint *)a ^ *(__global uint *)b;
	xi2 = 0;
	if (round == 6)
	  {
	    // skip padding byte
	    xi0 = (xi0 >> 8) | (xi1 << (64 - 8));
	    xi1 = (xi1 >> 8);
	  }
      }
    else if (round == 7 || round == 8)
      {
	// xor 8 bytes
	xi0 = half_aligned_long(a, 0) ^ half_aligned_long(b, 0);
	xi1 = 0;
	xi2 = 0;
	if (round == 8)
	  {
	    // skip padding byte
	    xi0 = (xi0 >> 8);
	  }
      }
    // invalid solutions (which start happenning in round 5) have duplicate
    // inputs and xor to zero, so discard them
    if (!xi0 && !xi1)
	return 0;
#else
#error "unsupported NR_ROWS_LOG"
#endif
    return ht_store(round, ht_dst, ENCODE_INPUTS(row, slot_a, slot_b),
	    xi0, xi1, xi2, 0, rowCounters);
}


/*
** Execute one Equihash round. Read from ht_src, XOR colliding pairs of Xi,
** store them in ht_dst.
*/
__device__
void equihash_round(uint round,
	__global char *ht_src,
	__global char *ht_dst,
	__global uint *debug,
	 uchar *first_words_data,
	 uint *collisionsData,
	 uint *collisionsNum,
	__global uint *rowCountersSrc,
	__global uint *rowCountersDst)
{
    uint		tid = get_global_id(0);
    uint		tlid = get_local_id(0);
    __global char	*p;
    uint		cnt;
    uchar	*first_words = &first_words_data[(NR_SLOTS)*tlid];
    uchar		mask;
    uint		i, j;
    // NR_SLOTS is already oversized (by a factor of OVERHEAD), but we want to
    // make it even larger
#ifdef ENABLE_DEBUG
	uint dropped_coll = 0;
	uint dropped_stor = 0;
#endif
	__global ulong	*a, *b;
    uint		xi_offset;
    // read first words of Xi from the previous (round - 1) hash table
    xi_offset = xi_offset_for_round(round - 1);
    // the mask is also computed to read data from the previous round
#if NR_ROWS_LOG == 16
    mask = ((!(round % 2)) ? 0x0f : 0xf0);
#elif NR_ROWS_LOG == 18
    mask = ((!(round % 2)) ? 0x03 : 0x30);
#elif NR_ROWS_LOG == 19
    mask = ((!(round % 2)) ? 0x01 : 0x10);
#elif NR_ROWS_LOG == 20
    mask = 0; /* we can vastly simplify the code below */
#else
#error "unsupported NR_ROWS_LOG"
#endif
    uint thCollNum = 0;
    *collisionsNum = 0;
    //barrier(CLK_LOCAL_MEM_FENCE);
    p = (ht_src + tid * NR_SLOTS * SLOT_LEN);
    uint rowIdx = tid/ROWS_PER_UINT;
    uint rowOffset = BITS_PER_ROW*(tid%ROWS_PER_UINT);
    cnt = (rowCountersSrc[rowIdx] >> rowOffset) & ROW_MASK;
    cnt = min(cnt, (uint)NR_SLOTS); // handle possible overflow in prev. round
    if (!cnt)
	// no elements in row, no collisions
	goto part2;
    p += xi_offset;
    for (i = 0; i < cnt; i++, p += SLOT_LEN)
	first_words[i] = (*(__global uchar *)p) & mask;
    // find collisions
    for (i = 0; i < cnt-1 && thCollNum < COLL_DATA_SIZE_PER_TH; i++)
      {
	uchar data_i = first_words[i];
	uint collision = (tid << 10) | (i << 5) | (i + 1);
	for (j = i+1; (j+4) < cnt;)
	  {
	      {
		uint isColl = ((data_i == first_words[j]) ? 1 : 0);
		if (isColl)
		  {
		    thCollNum++;
		    uint index = atomic_inc(collisionsNum);
		    collisionsData[index] = collision;
		  }
		collision++;
		j++;
	      }
	      {
		uint isColl = ((data_i == first_words[j]) ? 1 : 0);
		if (isColl)
		  {
		    thCollNum++;
		    uint index = atomic_inc(collisionsNum);
		    collisionsData[index] = collision;
		  }
		collision++;
		j++;
	      }
	      {
		uint isColl = ((data_i == first_words[j]) ? 1 : 0);
		if (isColl)
		  {
		    thCollNum++;
		    uint index = atomic_inc(collisionsNum);
		    collisionsData[index] = collision;
		  }
		collision++;
		j++;
	      }
	      {
		uint isColl = ((data_i == first_words[j]) ? 1 : 0);
		if (isColl)
		  {
		    thCollNum++;
		    uint index = atomic_inc(collisionsNum);
		    collisionsData[index] = collision;
		  }
		collision++;
		j++;
	      }
	  }
	for (; j < cnt; j++)
	  {
	    uint isColl = ((data_i == first_words[j]) ? 1 : 0);
	    if (isColl)
	      {
		thCollNum++;
		uint index = atomic_inc(collisionsNum);
		collisionsData[index] = collision;
	      }
	    collision++;
	  }
      }

part2:
    barrier(CLK_LOCAL_MEM_FENCE);
    uint totalCollisions = *collisionsNum;
    for (uint index = tlid; index < totalCollisions; index += get_local_size(0))
      {
	uint collision = collisionsData[index];
	uint collisionThreadId = collision >> 10;
	uint i = (collision >> 5) & 0x1F;
	uint j = collision & 0x1F;
	__global char *ptr = ht_src + collisionThreadId * NR_SLOTS * SLOT_LEN +
	    xi_offset;
	a = (__global ulong *)(ptr + i * SLOT_LEN);
	b = (__global ulong *)(ptr + j * SLOT_LEN);
	dropped_stor += xor_and_store(round, ht_dst, collisionThreadId, i, j,
		a, b, rowCountersDst);
	}
#ifdef ENABLE_DEBUG
	debug[tid * 2] = dropped_coll;
	debug[tid * 2 + 1] = dropped_stor;
#endif
}



/*
** This defines kernel_round1, kernel_round2, ..., kernel_round7.
*/
#define KERNEL_ROUND(N) \
__global__ void kernel_round ## N(__global char *ht_src, __global char *ht_dst, \
	__global uint *rowCountersSrc, __global uint *rowCountersDst, \
       	__global uint *debug) \
{ \
    __local uchar first_words_data[(NR_SLOTS)*THRD]; \
    __local uint    collisionsData[COLL_DATA_SIZE_PER_TH * THRD];\
    __local uint    collisionsNum; \
    equihash_round(N, ht_src, ht_dst, debug, first_words_data, collisionsData, \
	    &collisionsNum, rowCountersSrc, rowCountersDst); \
}
KERNEL_ROUND(1)
KERNEL_ROUND(2)
KERNEL_ROUND(3)
KERNEL_ROUND(4)
KERNEL_ROUND(5)
KERNEL_ROUND(6)
KERNEL_ROUND(7)


// kernel_round8 takes an extra argument, "sols"
__global__ 
void kernel_round8(__global char *ht_src, __global char *ht_dst,
	__global uint *rowCountersSrc, __global uint *rowCountersDst,
	__global uint *debug, __global sols_t *sols)
{
    uint		tid = get_global_id(0);
    __local uchar	first_words_data[(NR_SLOTS)*THRD];
    __local uint    collisionsData[COLL_DATA_SIZE_PER_TH * THRD];
    __local uint	collisionsNum;
    equihash_round(8, ht_src, ht_dst, debug, first_words_data, collisionsData,
	    &collisionsNum, rowCountersSrc, rowCountersDst);
    if (!tid)
	sols->nr = sols->likely_invalids = 0;
}


__device__ 
uint expand_ref(__global char *ht, uint xi_offset, uint row, uint slot)
{
    return *(__global uint *)(ht + row * NR_SLOTS * SLOT_LEN +
	    slot * SLOT_LEN + xi_offset - 4);
}


__device__ 
uint expand_refs(uint *ins, uint nr_inputs, __global char **htabs,
	uint round)
{
    __global char	*ht = htabs[round % 2];
    uint		i = nr_inputs - 1;
    uint		j = nr_inputs * 2 - 1;
    uint		xi_offset = xi_offset_for_round(round);
    int			dup_to_watch = -1;
    do
      {
	ins[j] = expand_ref(ht, xi_offset,
		DECODE_ROW(ins[i]), DECODE_SLOT1(ins[i]));
	ins[j - 1] = expand_ref(ht, xi_offset,
		DECODE_ROW(ins[i]), DECODE_SLOT0(ins[i]));
	if (!round)
	  {
	    if (dup_to_watch == -1)
		dup_to_watch = ins[j];
	    else if (ins[j] == dup_to_watch || ins[j - 1] == dup_to_watch)
		return 0;
	  }
	if (!i)
	    break ;
	i--;
	j -= 2;
      }
    while (1);
    return 1;
}


/*
** Verify if a potential solution is in fact valid.
*/
__device__ 
void potential_sol(__global char **htabs, __global sols_t *sols,
	uint ref0, uint ref1)
{
    uint	nr_values;
    uint	values_tmp[(1 << PARAM_K)];
    uint	sol_i;
    uint	i;
    nr_values = 0;
    values_tmp[nr_values++] = ref0;
    values_tmp[nr_values++] = ref1;
    uint round = PARAM_K - 1;
    do
      {
	round--;
	if (!expand_refs(values_tmp, nr_values, htabs, round))
	    return ;
	nr_values *= 2;
      }
    while (round > 0);
    // solution appears valid, copy it to sols
    sol_i = atomic_inc(&sols->nr);
    if (sol_i >= MAX_SOLS)
	return ;
    for (i = 0; i < (1 << PARAM_K); i++)
	sols->values[sol_i][i] = values_tmp[i];
    sols->valid[sol_i] = 1;
}


/*
** Scan the hash tables to find Equihash solutions.
*/
__global__
void kernel_sols(__global char *ht0, __global char *ht1, __global sols_t *sols,
	__global uint *rowCountersSrc, __global uint *rowCountersDst)
{
    uint		tid = get_global_id(0);
    __global char	*htabs[2] = { ht0, ht1 };
    __global char	*hcounters[2] = { (char *)rowCountersSrc, (char *)rowCountersDst };
    uint		ht_i = (PARAM_K - 1) % 2; // table filled at last round
    uint		cnt;
    uint		xi_offset = xi_offset_for_round(PARAM_K - 1);
    uint		i, j;
    __global char	*a, *b;
    uint		ref_i, ref_j;
    // it's ok for the collisions array to be so small, as if it fills up
    // the potential solutions are likely invalid (many duplicate inputs)
    ulong		collisions;
#if NR_ROWS_LOG >= 16 && NR_ROWS_LOG <= 20
    // in the final hash table, we are looking for a match on both the bits
    // part of the previous PREFIX colliding bits, and the last PREFIX bits.
    uint		mask = 0xffffff;
#else
#error "unsupported NR_ROWS_LOG"
#endif
    a = htabs[ht_i] + tid * NR_SLOTS * SLOT_LEN;
    uint rowIdx = tid/ROWS_PER_UINT;
    uint rowOffset = BITS_PER_ROW*(tid%ROWS_PER_UINT);
    cnt = (rowCountersSrc[rowIdx] >> rowOffset) & ROW_MASK;
    cnt = min(cnt, (uint)NR_SLOTS); // handle possible overflow in last round
    a += xi_offset;
    for (i = 0; i < cnt; i++, a += SLOT_LEN)
      {
	uint a_data = ((*(__global uint *)a) & mask);
	ref_i = *(__global uint *)(a - 4);
	for (j = i + 1, b = a + SLOT_LEN; j < cnt; j++, b += SLOT_LEN)
	  {
	    if (a_data == ((*(__global uint *)b) & mask))
	      {
		ref_j = *(__global uint *)(b - 4);
		collisions = ((ulong)ref_i << 32) | ref_j;
		goto exit1;
	      }
	  }
      }
    return;

exit1:
    potential_sol(htabs, sols, collisions >> 32, collisions & 0xffffffff);

}

struct __align__(64) c_context {
	char* buf_ht[2], *buf_sols, *buf_dbg;
	char *rowCounters[2];
	sols_t	*sols;
	u32 nthreads;
	size_t global_ws;


	c_context(const u32 n_threads) {
		nthreads = n_threads;
	}
	void* operator new(size_t i) {
		return _mm_malloc(i, 64);
	}
	void operator delete(void* p) {
		_mm_free(p);
	}
};



static size_t select_work_size_blake(void)
{
	size_t              work_size =
		64 * /* thread per wavefront */
		BLAKE_WPS * /* wavefront per simd */
		4 * /* simd per compute unit */
		36;
	// Make the work group size a multiple of the nr of wavefronts, while
	// dividing the number of inputs. This results in the worksize being a
	// power of 2.
	while (NR_INPUTS % work_size)
		work_size += 64;
	//debug("Blake: work size %zd\n", work_size);
	return work_size;
}

static void sort_pair(uint32_t *a, uint32_t len)
{
	uint32_t    *b = a + len;
	uint32_t     tmp, need_sorting = 0;
	for (uint32_t i = 0; i < len; i++)
		if (need_sorting || a[i] > b[i])
		{
			need_sorting = 1;
			tmp = a[i];
			a[i] = b[i];
			b[i] = tmp;
		}
		else if (a[i] < b[i])
			return;
}

static uint32_t verify_sol(sols_t *sols, unsigned sol_i)
{
	uint32_t  *inputs = sols->values[sol_i];
	uint32_t  seen_len = (1 << (PREFIX + 1)) / 8;
	uint8_t seen[(1 << (PREFIX + 1)) / 8];
	uint32_t  i;
	uint8_t tmp;
	// look for duplicate inputs
	memset(seen, 0, seen_len);
	for (i = 0; i < (1 << PARAM_K); i++)
	{
		tmp = seen[inputs[i] / 8];
		seen[inputs[i] / 8] |= 1 << (inputs[i] & 7);
		if (tmp == seen[inputs[i] / 8])
		{
			// at least one input value is a duplicate
			sols->valid[sol_i] = 0;
			return 0;
		}
	}
	// the valid flag is already set by the GPU, but set it again because
	// I plan to change the GPU code to not set it
	sols->valid[sol_i] = 1;
	// sort the pairs in place
	for (uint32_t level = 0; level < PARAM_K; level++)
		for (i = 0; i < (1 << PARAM_K); i += (2 << level))
			sort_pair(&inputs[i], 1 << level);
	return 1;
}

static void compress(uint8_t *out, uint32_t *inputs, uint32_t n)
{
	uint32_t byte_pos = 0;
	int32_t bits_left = PREFIX + 1;
	uint8_t x = 0;
	uint8_t x_bits_used = 0;
	uint8_t *pOut = out;
	while (byte_pos < n)
	{
		if (bits_left >= 8 - x_bits_used)
		{
			x |= inputs[byte_pos] >> (bits_left - 8 + x_bits_used);
			bits_left -= 8 - x_bits_used;
			x_bits_used = 8;
		}
		else if (bits_left > 0)
		{
			uint32_t mask = ~(-1 << (8 - x_bits_used));
			mask = ((~mask) >> bits_left) & mask;
			x |= (inputs[byte_pos] << (8 - x_bits_used - bits_left)) & mask;
			x_bits_used += bits_left;
			bits_left = 0;
		}
		else if (bits_left <= 0)
		{
			assert(!bits_left);
			byte_pos++;
			bits_left = PREFIX + 1;
		}
		if (x_bits_used == 8)
		{
			*pOut++ = x;
			x = x_bits_used = 0;
		}
	}
}

sa_cuda_context::sa_cuda_context(int tpb, int blocks, int id)
	: threadsperblock(tpb), totalblocks(blocks), device_id(id)
{
	checkCudaErrors(hipSetDevice(device_id));
	checkCudaErrors(hipDeviceReset());
	checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
	checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	eq = new c_context(threadsperblock * totalblocks);
#ifdef ENABLE_DEBUG
	size_t              dbg_size = NR_ROWS;
#else
	size_t              dbg_size = 1;
#endif

	checkCudaErrors(hipMalloc((void**)&eq->buf_dbg, dbg_size));
	checkCudaErrors(hipMalloc((void**)&eq->buf_ht[0], HT_SIZE));
	checkCudaErrors(hipMalloc((void**)&eq->buf_ht[1], HT_SIZE));
	checkCudaErrors(hipMalloc((void**)&eq->buf_sols, sizeof(sols_t)));
	checkCudaErrors(hipMalloc((void**)&eq->rowCounters[0], NR_ROWS));
	checkCudaErrors(hipMalloc((void**)&eq->rowCounters[1], NR_ROWS));

	eq->sols = (sols_t *)malloc(sizeof(*eq->sols));
}

sa_cuda_context::~sa_cuda_context()
{
	checkCudaErrors(hipSetDevice(device_id));
	checkCudaErrors(hipDeviceReset());
	delete eq;
}

void sa_cuda_context::solve(const char * tequihash_header, unsigned int tequihash_header_len, const char * nonce, unsigned int nonce_len, std::function<bool()> cancelf, std::function<void(const std::vector<uint32_t>&, size_t, const unsigned char*)> solutionf, std::function<void(void)> hashdonef)
{
	checkCudaErrors(hipSetDevice(device_id));

	unsigned char context[140];
	memset(context, 0, 140);
	memcpy(context, tequihash_header, tequihash_header_len);
	memcpy(context + tequihash_header_len, nonce, nonce_len);
	//printf("NR_SLOTS=%d NR_ROWS=%d\n",NR_SLOTS,NR_ROWS);
	c_context *miner = eq;
	
	//FUNCTION<<<totalblocks, threadsperblock>>>(ARGUMENTS)

	blake2b_state_t initialCtx;
	zcash_blake2b_init(&initialCtx, ZCASH_HASH_LEN, PARAM_N, PARAM_K);
	zcash_blake2b_update(&initialCtx, (const uint8_t*)context, 128, 0);

	void* buf_blake_st;
	checkCudaErrors(hipMalloc((void**)&buf_blake_st, sizeof(blake2b_state_s)));
	checkCudaErrors(hipMemcpy(buf_blake_st, &initialCtx, sizeof(blake2b_state_s), hipMemcpyHostToDevice));
	
	for (unsigned round = 0; round < PARAM_K; round++) {
//		if (round < 2) {
			//every round
			checkCudaErrors(hipMemset(miner->rowCounters[round % 2],0,NR_ROWS));
//			kernel_init_ht<<<NR_ROWS / ROWS_PER_UINT / 256, 256>>>(miner->buf_ht[round & 1],(uint*)miner->rowCounters[round % 2]);
//			printf("%d %d %d\n",NR_ROWS / ROWS_PER_UINT / 256, NR_ROWS,ROWS_PER_UINT);
//			exit(-1);
//		}
		if (!round)	{
			miner->global_ws = select_work_size_blake();
		} else {
			miner->global_ws = NR_ROWS;
		}
		// cancel function
		switch (round) {
		case 0:
			kernel_round0<<<NR_INPUTS/THRD,THRD>>>((ulong*)buf_blake_st, miner->buf_ht[round & 1], (uint*)miner->rowCounters[round % 2],(uint*)miner->buf_dbg);
//			printf("%d %d\n",totalblocks, NR_INPUTS);
//                        exit(-1);
			break;
		case 1:
			kernel_round1<<<BLK,THRD>>>(miner->buf_ht[(round - 1) & 1], miner->buf_ht[round & 1],(uint*)miner->rowCounters[(round - 1) % 2],(uint*)miner->rowCounters[round % 2], (uint*)miner->buf_dbg);
			//exit(0);
			break;
		case 2:
			kernel_round2<<<BLK,THRD>>>(miner->buf_ht[(round - 1) & 1], miner->buf_ht[round & 1], (uint*)miner->rowCounters[(round - 1) % 2],(uint*)miner->rowCounters[round % 2], (uint*)miner->buf_dbg);
//			exit(0);
			break;
		case 3:
			kernel_round3<<<BLK,THRD>>>(miner->buf_ht[(round - 1) & 1], miner->buf_ht[round & 1],(uint*)miner->rowCounters[(round - 1) % 2],(uint*)miner->rowCounters[round % 2], (uint*)miner->buf_dbg);
			break;
		case 4:
			kernel_round4<<<BLK,THRD>>>(miner->buf_ht[(round - 1) & 1], miner->buf_ht[round & 1],(uint*)miner->rowCounters[(round - 1) % 2],(uint*)miner->rowCounters[round % 2], (uint*)miner->buf_dbg);
			break;
		case 5:
			kernel_round5<<<BLK,THRD>>>(miner->buf_ht[(round - 1) & 1], miner->buf_ht[round & 1],(uint*)miner->rowCounters[(round - 1) % 2],(uint*)miner->rowCounters[round % 2], (uint*)miner->buf_dbg);
//exit(0);
			break;
		case 6:
			kernel_round6<<<BLK,THRD>>>(miner->buf_ht[(round - 1) & 1], miner->buf_ht[round & 1],(uint*)miner->rowCounters[(round - 1) % 2],(uint*)miner->rowCounters[round % 2], (uint*)miner->buf_dbg);
	//	exit(0);
			break;
		case 7:
			kernel_round7<<<BLK,THRD>>>(miner->buf_ht[(round - 1) & 1], miner->buf_ht[round & 1],(uint*)miner->rowCounters[(round - 1) % 2],(uint*)miner->rowCounters[round % 2], (uint*)miner->buf_dbg);
//			exit(0);
			break;
		case 8:
			kernel_round8<<<BLK,THRD>>>(miner->buf_ht[(round - 1) & 1], miner->buf_ht[round & 1],(uint*)miner->rowCounters[(round - 1) % 2],(uint*)miner->rowCounters[round % 2], (uint*)miner->buf_dbg, (sols_t*)miner->buf_sols);
			break;
		}
		if (cancelf()) return;
	}
	kernel_sols<<<NR_ROWS/32,32>>>(miner->buf_ht[0], miner->buf_ht[1], (sols_t*)miner->buf_sols,(uint*)miner->rowCounters[0],(uint*)miner->rowCounters[1]);

	checkCudaErrors(hipMemcpy(miner->sols, miner->buf_sols, sizeof(*miner->sols), hipMemcpyDeviceToHost));

	if (miner->sols->nr > MAX_SOLS)
		miner->sols->nr = MAX_SOLS;

	for (unsigned sol_i = 0; sol_i < miner->sols->nr; sol_i++) {
		verify_sol(miner->sols, sol_i);
	}

	uint8_t proof[COMPRESSED_PROOFSIZE * 2];
	for (uint32_t i = 0; i < miner->sols->nr; i++) {
		if (miner->sols->valid[i]) {
			compress(proof, (uint32_t *)(miner->sols->values[i]), 1 << PARAM_K);
			solutionf(std::vector<uint32_t>(0), 1344, proof);
		}
	}
	hashdonef();
}
