#include "hip/hip_runtime.h"
/*
  Equihash solver created by djeZo (l33tsoftw@gmail.com) for NiceHash

  Based on CUDA solver by John Tromp released under MIT license.

  Some helper functions taken out of OpenCL solver by Marc Bevand
  released under MIT license.

  cuda_djezo solver is released by NiceHash (www.nicehash.com) under
  GPL 3.0 license. If you don't have a copy, you can obtain one from
  https://www.gnu.org/licenses/gpl-3.0.txt
*/

/*
The MIT License (MIT)

Copyright (c) 2016 John Tromp

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software, and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

/*
The MIT License (MIT)

Copyright (c) 2016 Marc Bevand

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software, and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#ifdef WIN32
#include <Windows.h>
#endif
#include <stdint.h>
#include <string.h>
#include <stdlib.h>
#include <stdbool.h>
#include <stdio.h>
#include <assert.h>
#include <functional>
#include <vector>
#include <iostream>
#include <mutex>

#include "eqcuda.hpp"
#include "sm_32_intrinsics.h"

#define WN	200
#define WK	9
#define NDIGITS		(WK+1)
#define DIGITBITS	(WN/(NDIGITS))
#define PROOFSIZE (1<<WK)
#define BASE (1<<DIGITBITS)
#define NHASHES (2*BASE)
#define HASHESPERBLAKE (512/WN)
#define HASHOUT (HASHESPERBLAKE*WN/8)
#define NBLOCKS ((NHASHES + HASHESPERBLAKE - 1) / HASHESPERBLAKE)
#define BUCKBITS (DIGITBITS - RB)
#define NBUCKETS (1 << BUCKBITS)
#define BUCKMASK (NBUCKETS - 1)
#define SLOTBITS (RB + 2)
#define SLOTRANGE (1 << SLOTBITS)
#define NSLOTS SM
#define SLOTMASK (SLOTRANGE - 1)
#define NRESTS (1 << RB)
#define RESTMASK (NRESTS - 1)
#define CANTORBITS (2 * SLOTBITS - 2)
#define CANTORMASK ((1 << CANTORBITS) - 1)
#define CANTORMAXSQRT (2 * NSLOTS)
#define RB8_NSLOTS 640
#define RB8_NSLOTS_LD 624
#define FD_THREADS 128

// reduce vstudio warnings (__byteperm, blockIdx...)
#ifdef __INTELLISENSE__
#include <hip/device_functions.h>
#include <>
#define __launch_bounds__(max_tpb, min_blocks)
#define __CUDA_ARCH__ 520
uint32_t __byte_perm(uint32_t x, uint32_t y, uint32_t z);
uint32_t __byte_perm(uint32_t x, uint32_t y, uint32_t z);
uint32_t __shfl(uint32_t x, uint32_t y, uint32_t z);
uint32_t atomicExch(uint32_t *x, uint32_t y);
uint32_t atomicAdd(uint32_t *x, uint32_t y);
void __syncthreads(void);
void __threadfence(void);
void __threadfence_block(void);
uint32_t __ldg(const uint32_t* address);
uint64_t __ldg(const uint64_t* address);
uint4 __ldca(const uint4 *ptr);
u32 __ldca(const u32 *ptr);
u32 umin(const u32, const u32);
u32 umax(const u32, const u32);
#endif


typedef u32 proof[PROOFSIZE];


struct __align__(32) slot
{
	u32 hash[8];
};


struct __align__(16) slotsmall
{
	u32 hash[4];
};


struct __align__(8) slottiny
{
	u32 hash[2];
};


template <u32 RB, u32 SM>
struct equi
{
	slot round0trees[4096][RB8_NSLOTS];
	slot trees[1][NBUCKETS][NSLOTS];
	struct
	{
		slotsmall treessmall[NSLOTS];
		slottiny treestiny[NSLOTS];
	} round2trees[NBUCKETS];
	struct
	{
		slotsmall treessmall[NSLOTS];
		slottiny treestiny[NSLOTS];
	} round3trees[NBUCKETS];
	slotsmall treessmall[4][NBUCKETS][NSLOTS];
	slottiny treestiny[1][4096][RB8_NSLOTS_LD];
	u32 round4bidandsids[NBUCKETS][NSLOTS];
	union
	{
		u64 blake_h[8];
		u32 blake_h32[16];
	};
	struct
	{
		u32 nslots8[4096];
		u32 nslots0[4096];
		u32 nslots[9][NBUCKETS];
		scontainerreal srealcont;
	} edata;
};


__device__ __constant__ const u64 blake_iv[] =
{
	0x6a09e667f3bcc908, 0xbb67ae8584caa73b,
	0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1,
	0x510e527fade682d1, 0x9b05688c2b3e6c1f,
	0x1f83d9abfb41bd6b, 0x5be0cd19137e2179,
};

__device__ __forceinline__ uint2 operator^ (uint2 a, uint2 b)
{
	return make_uint2(a.x ^ b.x, a.y ^ b.y);
}

__device__ __forceinline__ uint4 operator^ (uint4 a, uint4 b)
{
	return make_uint4(a.x ^ b.x, a.y ^ b.y, a.z ^ b.z, a.w ^ b.w);
}

__device__ __forceinline__ uint2 ROR2(const uint2 a, const int offset) 
{
	uint2 result;
	{
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
	}
	return result;
}

__device__ __forceinline__ uint2 SWAPUINT2(uint2 value) 
{
	return make_uint2(value.y, value.x);
}

__device__ __forceinline__ uint2 ROR24(const uint2 a)
{
	uint2 result;
	result.x = __byte_perm(a.y, a.x, 0x2107);
	result.y = __byte_perm(a.y, a.x, 0x6543);
	return result;
}

__device__ __forceinline__ uint2 ROR16(const uint2 a)
{
	uint2 result;
	result.x = __byte_perm(a.y, a.x, 0x1076);
	result.y = __byte_perm(a.y, a.x, 0x5432);
	return result;
}

__device__ __forceinline__ void G2(u64 & a, u64 & b, u64 & c, u64 & d, u64 x, u64 y) 
{
	a = a + b + x;
	((uint2*)&d)[0] = SWAPUINT2(((uint2*)&d)[0] ^ ((uint2*)&a)[0]);
	c = c + d;
	((uint2*)&b)[0] = ROR24(((uint2*)&b)[0] ^ ((uint2*)&c)[0]);
	a = a + b + y;
	((uint2*)&d)[0] = ROR16(((uint2*)&d)[0] ^ ((uint2*)&a)[0]);
	c = c + d;
	((uint2*)&b)[0] = ROR2(((uint2*)&b)[0] ^ ((uint2*)&c)[0], 63U);
}


struct packer_default
{
	__device__ __forceinline__ static u32 set_bucketid_and_slots(const u32 bucketid, const u32 s0, const u32 s1, const u32 RB, const u32 SM)
	{
		return (((bucketid << SLOTBITS) | s0) << SLOTBITS) | s1;
	}

	__device__ __forceinline__ static u32 get_bucketid(const u32 bid, const u32 RB, const u32 SM)
	{
		// BUCKMASK-ed to prevent illegal memory accesses in case of memory errors
		return (bid >> (2 * SLOTBITS)) & BUCKMASK;
	}

	__device__ __forceinline__ static u32 get_slot0(const u32 bid, const u32 s1, const u32 RB, const u32 SM)
	{
		return bid & SLOTMASK;
	}

	__device__ __forceinline__ static u32 get_slot1(const u32 bid, const u32 RB, const u32 SM)
	{
		return (bid >> SLOTBITS) & SLOTMASK;
	}
};


struct packer_cantor
{
	__device__ __forceinline__ static u32 cantor(const u32 s0, const u32 s1)
	{
		u32 a = umax(s0, s1);
		u32 b = umin(s0, s1);
		return a * (a + 1) / 2 + b;
	}

	__device__ __forceinline__ static u32 set_bucketid_and_slots(const u32 bucketid, const u32 s0, const u32 s1, const u32 RB, const u32 SM)
	{
		return (bucketid << CANTORBITS) | cantor(s0, s1);
	}

	__device__ __forceinline__ static u32 get_bucketid(const u32 bid, const u32 RB, const u32 SM)
	{
		return (bid >> CANTORBITS) & BUCKMASK;
	}

	__device__ __forceinline__ static u32 get_slot0(const u32 bid, const u32 s1, const u32 RB, const u32 SM)
	{
		return ((bid & CANTORMASK) - cantor(0, s1)) & SLOTMASK;
	}

	__device__ __forceinline__ static u32 get_slot1(const u32 bid, const u32 RB, const u32 SM)
	{
		u32 k, q, sqr = 8 * (bid & CANTORMASK) + 1;
		// this k=sqrt(sqr) computing loop averages 3.4 iterations out of maximum 9
		for (k = CANTORMAXSQRT; (q = sqr / k) < k; k = (k + q) / 2);
		return ((k - 1) / 2) & SLOTMASK;
	}
};


template <u32 RB, u32 SM, typename PACKER>
__global__ void digit_first(equi<RB, SM>* eq, u32 nonce)
{
	const u32 block = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ u64 hash_h[8];
	u32* hash_h32 = (u32*)hash_h;

	if (threadIdx.x < 16)
		hash_h32[threadIdx.x] = __ldca(&eq->blake_h32[threadIdx.x]);

	__syncthreads();

	u64 m = (u64)block << 32 | (u64)nonce;

	union
	{
		u64 v[16];
		u32 v32[32];
		uint4 v128[8];
	};

	v[0] = hash_h[0];
	v[1] = hash_h[1];
	v[2] = hash_h[2];
	v[3] = hash_h[3];
	v[4] = hash_h[4];
	v[5] = hash_h[5];
	v[6] = hash_h[6];
	v[7] = hash_h[7];
	v[8] = blake_iv[0];
	v[9] = blake_iv[1];
	v[10] = blake_iv[2];
	v[11] = blake_iv[3];
	v[12] = blake_iv[4] ^ (128 + 16);
	v[13] = blake_iv[5];
	v[14] = blake_iv[6] ^ 0xffffffffffffffff;
	v[15] = blake_iv[7];

	// mix 1
	G2(v[0], v[4], v[8], v[12], 0, m);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 2
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], m, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 3
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, m);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 4
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, m);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 5
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, m);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 6
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], m, 0);

	// mix 7
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], m, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 8
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, m);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 9
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], m, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 10
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], m, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 11
	G2(v[0], v[4], v[8], v[12], 0, m);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 12
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], m, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	v[0] ^= hash_h[0] ^ v[8];
	v[1] ^= hash_h[1] ^ v[9];
	v[2] ^= hash_h[2] ^ v[10];
	v[3] ^= hash_h[3] ^ v[11];
	v[4] ^= hash_h[4] ^ v[12];
	v[5] ^= hash_h[5] ^ v[13];
	v32[12] ^= hash_h32[12] ^ v32[28];

	u32 bexor = __byte_perm(v32[0], 0, 0x4012); // first 20 bits
	u32 bucketid;
	asm("bfe.u32 %0, %1, 12, 12;" : "=r"(bucketid) : "r"(bexor));
	u32 slotp = atomicAdd(&eq->edata.nslots0[bucketid], 1);
	if (slotp < RB8_NSLOTS)
	{
		slot* s = &eq->round0trees[bucketid][slotp];

		uint4 tt;
		tt.x = __byte_perm(v32[0], v32[1], 0x1234);
		tt.y = __byte_perm(v32[1], v32[2], 0x1234);
		tt.z = __byte_perm(v32[2], v32[3], 0x1234);
		tt.w = __byte_perm(v32[3], v32[4], 0x1234);
		*(uint4*)(&s->hash[0]) = tt;

		tt.x = __byte_perm(v32[4], v32[5], 0x1234);
		tt.y = __byte_perm(v32[5], v32[6], 0x1234);
		tt.z = 0;
		tt.w = block << 1;
		*(uint4*)(&s->hash[4]) = tt;
	}

	bexor = __byte_perm(v32[6], 0, 0x0123);
	asm("bfe.u32 %0, %1, 12, 12;" : "=r"(bucketid) : "r"(bexor));
	slotp = atomicAdd(&eq->edata.nslots0[bucketid], 1);
	if (slotp < RB8_NSLOTS)
	{
		slot* s = &eq->round0trees[bucketid][slotp];

		uint4 tt;
		tt.x = __byte_perm(v32[6], v32[7], 0x2345);
		tt.y = __byte_perm(v32[7], v32[8], 0x2345);
		tt.z = __byte_perm(v32[8], v32[9], 0x2345);
		tt.w = __byte_perm(v32[9], v32[10], 0x2345);
		*(uint4*)(&s->hash[0]) = tt;

		tt.x = __byte_perm(v32[10], v32[11], 0x2345);
		tt.y = __byte_perm(v32[11], v32[12], 0x2345);
		tt.z = 0;
		tt.w = (block << 1) + 1;
		*(uint4*)(&s->hash[4]) = tt;
	}
}

/*
  Functions digit_1 to digit_8 works by the same principle;
  Each thread does 2-3 slot loads (loads are coalesced). 
  Xorwork of slots is loaded into shared memory and is kept in registers (except for digit_1).
  At the same time, restbits (8 or 9 bits) in xorwork are used for collisions. 
  Restbits determine position in ht.
  Following next is pair creation. First one (or two) pairs' xorworks are put into global memory
  as soon as possible, the rest pairs are saved in shared memory (one u32 per pair - 16 bit indices). 
  In most cases, all threads have one (or two) pairs so with this trick, we offload memory writes a bit in last step.
  In last step we save xorwork of pairs in memory.
*/
template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_1(equi<RB, SM>* eq)
{
	__shared__ u16 ht[256][SSM - 1];
	__shared__ uint2 lastword1[RB8_NSLOTS];
	__shared__ uint4 lastword2[RB8_NSLOTS];
	__shared__ int ht_len[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	if (threadid < 256)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;
	else if (threadid == (THREADS - 33))
		next_pair = 0;

	u32 bsize = umin(eq->edata.nslots0[bucketid], RB8_NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	uint2 ta[2];
	uint4 tb[2];

	u32 si[2];

	// enable this to make fully safe shared mem operations;
	// disabled gains some speed, but can rarely cause a crash
	//__syncthreads();

#pragma unroll
	for (u32 i = 0; i != 2; ++i)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

		const slot* pslot1 = eq->round0trees[bucketid] + si[i];

		// get xhash
		uint4 a1 = *(uint4*)(&pslot1->hash[0]);
		uint2 a2 = *(uint2*)(&pslot1->hash[4]);
		ta[i].x = a1.x;
		ta[i].y = a1.y;
		lastword1[si[i]] = ta[i];
		tb[i].x = a1.z;
		tb[i].y = a1.w;
		tb[i].z = a2.x;
		tb[i].w = a2.y;
		lastword2[si[i]] = tb[i];

		asm("bfe.u32 %0, %1, 20, 8;" : "=r"(hr[i]) : "r"(ta[i].x));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();
	int* pairs = ht_len;

	u32 xors[6];
	u32 xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i != 2; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			*(uint2*)(&xors[0]) = ta[i] ^ lastword1[p];

			asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(RB), "r"(BUCKBITS));
			xorslot = atomicAdd(&eq->edata.nslots[1][xorbucketid], 1);

			if (xorslot < NSLOTS)
			{
				*(uint4*)(&xors[2]) = lastword2[si[i]] ^ lastword2[p];

				slot &xs = eq->trees[0][xorbucketid][xorslot];
				*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[1]);
				uint4 ttx;
				ttx.x = xors[5];
				ttx.y = xors[0];
				ttx.z = packer_default::set_bucketid_and_slots(bucketid, si[i], p, 8, RB8_NSLOTS);
				ttx.w = 0;
				*(uint4*)(&xs.hash[4]) = ttx;
			}

			for (int k = 1; k != pos[i]; ++k)
			{
				u32 pindex = atomicAdd(&pairs_len, 1);
				if (pindex >= MAXPAIRS) break;
				u16 prev = ht[hr[i]][k];
				pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);

	u32 i, k;
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		*(uint2*)(&xors[0]) = lastword1[i] ^ lastword1[k];

		asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(RB), "r"(BUCKBITS));
		xorslot = atomicAdd(&eq->edata.nslots[1][xorbucketid], 1);

		if (xorslot < NSLOTS)
		{
			*(uint4*)(&xors[2]) = lastword2[i] ^ lastword2[k];

			slot &xs = eq->trees[0][xorbucketid][xorslot];
			*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[1]);
			uint4 ttx;
			ttx.x = xors[5];
			ttx.y = xors[0];
			ttx.z = packer_default::set_bucketid_and_slots(bucketid, i, k, 8, RB8_NSLOTS);
			ttx.w = 0;
			*(uint4*)(&xs.hash[4]) = ttx;
		}
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_2(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][SSM - 1];
	__shared__ u32 lastword1[NSLOTS];
	__shared__ uint4 lastword2[NSLOTS];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	if (threadid < NRESTS)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;
	else if (threadid == (THREADS - 33))
		next_pair = 0;

	slot* buck = eq->trees[0][bucketid];
	u32 bsize = umin(eq->edata.nslots[1][bucketid], NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	u32 ta[2];
	uint4 tt[2];

	u32 si[2];

	// enable this to make fully safe shared mem operations;
	// disabled gains some speed, but can rarely cause a crash
	//__syncthreads();

#pragma unroll
	for (u32 i = 0; i != 2; ++i)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

		// get slot
		const slot* pslot1 = buck + si[i];

		uint4 ttx = *(uint4*)(&pslot1->hash[0]);
		lastword1[si[i]] = ta[i] = ttx.x;
		uint2 tty = *(uint2*)(&pslot1->hash[4]);
		tt[i].x = ttx.y;
		tt[i].y = ttx.z;
		tt[i].z = ttx.w;
		tt[i].w = tty.x;
		lastword2[si[i]] = tt[i];

		hr[i] = tty.y & RESTMASK;
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();

	u32 xors[5];
	u32 xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i != 2; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			xors[0] = ta[i] ^ lastword1[p];

			xorbucketid = xors[0] >> (12 + RB);
			xorslot = atomicAdd(&eq->edata.nslots[2][xorbucketid], 1);
			if (xorslot < NSLOTS)
			{
				*(uint4*)(&xors[1]) = tt[i] ^ lastword2[p];
				slotsmall &xs = eq->round2trees[xorbucketid].treessmall[xorslot];
				*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[0]);
				slottiny &xst = eq->round2trees[xorbucketid].treestiny[xorslot];
				uint2 ttx;
				ttx.x = xors[4];
				ttx.y = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
				*(uint2*)(&xst.hash[0]) = ttx;
			}

			for (int k = 1; k != pos[i]; ++k)
			{
				u32 pindex = atomicAdd(&pairs_len, 1);
				if (pindex >= MAXPAIRS) break;
				u16 prev = ht[hr[i]][k];
				pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);

	u32 i, k;
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		xors[0] = lastword1[i] ^ lastword1[k];

		xorbucketid = xors[0] >> (12 + RB);
		xorslot = atomicAdd(&eq->edata.nslots[2][xorbucketid], 1);
		if (xorslot < NSLOTS)
		{
			*(uint4*)(&xors[1]) = lastword2[i] ^ lastword2[k];
			slotsmall &xs = eq->round2trees[xorbucketid].treessmall[xorslot];
			*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[0]);
			slottiny &xst = eq->round2trees[xorbucketid].treestiny[xorslot];
			uint2 ttx;
			ttx.x = xors[4];
			ttx.y = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
			*(uint2*)(&xst.hash[0]) = ttx;
		}
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_3(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ uint4 lastword1[NSLOTS];
	__shared__ u32 lastword2[NSLOTS];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	if (threadid < NRESTS)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;
	else if (threadid == (THREADS - 33))
		next_pair = 0;

	u32 bsize = umin(eq->edata.nslots[2][bucketid], NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	u32 si[2];
	uint4 tt[2];
	u32 ta[2];

	// enable this to make fully safe shared mem operations;
	// disabled gains some speed, but can rarely cause a crash
	//__syncthreads();

#pragma unroll
	for (u32 i = 0; i != 2; ++i)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

		slotsmall &xs = eq->round2trees[bucketid].treessmall[si[i]];
		slottiny &xst = eq->round2trees[bucketid].treestiny[si[i]];

		tt[i] = *(uint4*)(&xs.hash[0]);
		lastword1[si[i]] = tt[i];
		ta[i] = xst.hash[0];
		lastword2[si[i]] = ta[i];
		asm("bfe.u32 %0, %1, 12, %2;" : "=r"(hr[i]) : "r"(tt[i].x), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();

	u32 xors[5];
	u32 bexor, xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i != 2; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			xors[4] = ta[i] ^ lastword2[p];

			if (xors[4] != 0)
			{
				*(uint4*)(&xors[0]) = tt[i] ^ lastword1[p];

				bexor = __byte_perm(xors[0], xors[1], 0x2107);
				asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(bexor), "r"(RB), "r"(BUCKBITS));
				xorslot = atomicAdd(&eq->edata.nslots[3][xorbucketid], 1);

				if (xorslot < NSLOTS)
				{
					slotsmall &xs = eq->round3trees[xorbucketid].treessmall[xorslot];
					*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[1]);
					slottiny &xst = eq->round3trees[xorbucketid].treestiny[xorslot];
					uint2 ttx;
					ttx.x = bexor;
					ttx.y = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
					*(uint2*)(&xst.hash[0]) = ttx;
				}
			}

			for (int k = 1; k != pos[i]; ++k)
			{
				u32 pindex = atomicAdd(&pairs_len, 1);
				if (pindex >= MAXPAIRS) break;
				u16 prev = ht[hr[i]][k];
				pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);

	u32 i, k;
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		xors[4] = lastword2[i] ^ lastword2[k];

		if (xors[4] != 0)
		{
			*(uint4*)(&xors[0]) = lastword1[i] ^ lastword1[k];

			bexor = __byte_perm(xors[0], xors[1], 0x2107);
			asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(bexor), "r"(RB), "r"(BUCKBITS));
			xorslot = atomicAdd(&eq->edata.nslots[3][xorbucketid], 1);

			if (xorslot < NSLOTS)
			{
				slotsmall &xs = eq->round3trees[xorbucketid].treessmall[xorslot];
				*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[1]);
				slottiny &xst = eq->round3trees[xorbucketid].treestiny[xorslot];
				uint2 ttx;
				ttx.x = bexor;
				ttx.y = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
				*(uint2*)(&xst.hash[0]) = ttx;
			}
		}
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_4(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ uint4 lastword[NSLOTS];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	if (threadid < NRESTS)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;
	else if (threadid == (THREADS - 33))
		next_pair = 0;

	u32 bsize = umin(eq->edata.nslots[3][bucketid], NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	u32 si[2];
	uint4 tt[2];

	// enable this to make fully safe shared mem operations;
	// disabled gains some speed, but can rarely cause a crash
	//__syncthreads();

#pragma unroll
	for (u32 i = 0; i != 2; ++i)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

		slotsmall &xs = eq->round3trees[bucketid].treessmall[si[i]];
		slottiny &xst = eq->round3trees[bucketid].treestiny[si[i]];

		// get xhash
		tt[i] = *(uint4*)(&xs.hash[0]);
		lastword[si[i]] = tt[i];
		hr[i] = xst.hash[0] & RESTMASK;
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();
	u32 xors[4];
	u32 xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i != 2; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			*(uint4*)(&xors[0]) = tt[i] ^ lastword[p];

			if (xors[3] != 0)
			{
				asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(4 + RB), "r"(BUCKBITS));
				xorslot = atomicAdd(&eq->edata.nslots[4][xorbucketid], 1);
				if (xorslot < NSLOTS)
				{
					slotsmall &xs = eq->treessmall[3][xorbucketid][xorslot];
					*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[0]);

					eq->round4bidandsids[xorbucketid][xorslot] = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
				}
			}

			for (int k = 1; k != pos[i]; ++k)
			{
				u32 pindex = atomicAdd(&pairs_len, 1);
				if (pindex >= MAXPAIRS) break;
				u16 prev = ht[hr[i]][k];
				pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);
	u32 i, k;
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		*(uint4*)(&xors[0]) = lastword[i] ^ lastword[k];
		if (xors[3] != 0)
		{
			asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(4 + RB), "r"(BUCKBITS));
			xorslot = atomicAdd(&eq->edata.nslots[4][xorbucketid], 1);
			if (xorslot < NSLOTS)
			{
				slotsmall &xs = eq->treessmall[3][xorbucketid][xorslot];
				*(uint4*)(&xs.hash[0]) = *(uint4*)(&xors[0]);
				eq->round4bidandsids[xorbucketid][xorslot] = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
			}
		}
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS, u32 THREADS>
__global__ void digit_5(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ uint4 lastword[NSLOTS];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	if (threadid < NRESTS)
		ht_len[threadid] = 0;
	else if (threadid == (THREADS - 1))
		pairs_len = 0;
	else if (threadid == (THREADS - 33))
		next_pair = 0;

	slotsmall* buck = eq->treessmall[3][bucketid];
	u32 bsize = umin(eq->edata.nslots[4][bucketid], NSLOTS);

	u32 hr[2];
	int pos[2];
	pos[0] = pos[1] = SSM;

	u32 si[2];
	uint4 tt[2];

	// enable this to make fully safe shared mem operations;
	// disabled gains some speed, but can rarely cause a crash
	//__syncthreads();

#pragma unroll
	for (u32 i = 0; i != 2; ++i)
	{
		si[i] = i * THREADS + threadid;
		if (si[i] >= bsize) break;

		const slotsmall* pslot1 = buck + si[i];

		tt[i] = *(uint4*)(&pslot1->hash[0]);
		lastword[si[i]] = tt[i];
		asm("bfe.u32 %0, %1, 4, %2;" : "=r"(hr[i]) : "r"(tt[i].x), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();
	u32 xors[4];
	u32 bexor, xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i != 2; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			*(uint4*)(&xors[0]) = tt[i] ^ lastword[p];

			if (xors[3] != 0)
			{
				bexor = __byte_perm(xors[0], xors[1], 0x1076);
				asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(bexor), "r"(RB), "r"(BUCKBITS));
				xorslot = atomicAdd(&eq->edata.nslots[5][xorbucketid], 1);
				if (xorslot < NSLOTS)
				{
					slotsmall &xs = eq->treessmall[2][xorbucketid][xorslot];
					uint4 ttx;
					ttx.x = xors[1];
					ttx.y = xors[2];
					ttx.z = xors[3];
					ttx.w = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
					*(uint4*)(&xs.hash[0]) = ttx;
				}
			}

			for (int k = 1; k != pos[i]; ++k)
			{
				u32 pindex = atomicAdd(&pairs_len, 1);
				if (pindex >= MAXPAIRS) break;
				u16 prev = ht[hr[i]][k];
				pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);
	u32 i, k;
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		int pair = pairs[s];
		i = __byte_perm(pair, 0, 0x4510);
		k = __byte_perm(pair, 0, 0x4532);

		*(uint4*)(&xors[0]) = lastword[i] ^ lastword[k];

		if (xors[3] != 0)
		{
			bexor = __byte_perm(xors[0], xors[1], 0x1076);
			asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(bexor), "r"(RB), "r"(BUCKBITS));
			xorslot = atomicAdd(&eq->edata.nslots[5][xorbucketid], 1);
			if (xorslot < NSLOTS)
			{
				slotsmall &xs = eq->treessmall[2][xorbucketid][xorslot];
				uint4 tt;
				tt.x = xors[1];
				tt.y = xors[2];
				tt.z = xors[3];
				tt.w = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
				*(uint4*)(&xs.hash[0]) = tt;
			}
		}
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS>
__global__ void digit_6(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ uint2 lastword1[NSLOTS];
	__shared__ u32 lastword2[NSLOTS];
	__shared__ int ht_len[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 bsize_sh;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	ht_len[threadid] = 0;
	if (threadid == (NRESTS - 1))
	{
		pairs_len = 0;
		next_pair = 0;
	}
	else if (threadid == (NRESTS - 33))
		bsize_sh = umin(eq->edata.nslots[5][bucketid], NSLOTS);

	slotsmall* buck = eq->treessmall[2][bucketid];

	u32 hr[3];
	int pos[3];
	pos[0] = pos[1] = pos[2] = SSM;

	u32 si[3];
	uint4 tt[3];

	__syncthreads();

	u32 bsize = bsize_sh;

#pragma unroll
	for (u32 i = 0; i != 3; ++i)
	{
		si[i] = i * NRESTS + threadid;
		if (si[i] >= bsize) break;

		const slotsmall* pslot1 = buck + si[i];

		tt[i] = *(uint4*)(&pslot1->hash[0]);
		lastword1[si[i]] = *(uint2*)(&tt[i].x);
		lastword2[si[i]] = tt[i].z;
		asm("bfe.u32 %0, %1, 16, %2;" : "=r"(hr[i]) : "r"(tt[i].x), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	// doing this to save shared memory
	int* pairs = ht_len;
	__syncthreads();

	u32 xors[3];
	u32 bexor, xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i != 3; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			xors[2] = tt[i].z ^ lastword2[p];

			if (xors[2] != 0)
			{
				*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ lastword1[p];

				bexor = __byte_perm(xors[0], xors[1], 0x1076);
				xorbucketid = bexor >> (12 + RB);
				xorslot = atomicAdd(&eq->edata.nslots[6][xorbucketid], 1);
				if (xorslot < NSLOTS)
				{
					slotsmall &xs = eq->treessmall[0][xorbucketid][xorslot];
					uint4 ttx;
					ttx.x = xors[1];
					ttx.y = xors[2];
					ttx.z = bexor;
					ttx.w = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
					*(uint4*)(&xs.hash[0]) = ttx;
				}
			}

			if (pos[i] > 1)
			{
				p = ht[hr[i]][1];

				xors[2] = tt[i].z ^ lastword2[p];

				if (xors[2] != 0)
				{
					*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ lastword1[p];

					bexor = __byte_perm(xors[0], xors[1], 0x1076);
					xorbucketid = bexor >> (12 + RB);
					xorslot = atomicAdd(&eq->edata.nslots[6][xorbucketid], 1);
					if (xorslot < NSLOTS)
					{
						slotsmall &xs = eq->treessmall[0][xorbucketid][xorslot];
						uint4 ttx;
						ttx.x = xors[1];
						ttx.y = xors[2];
						ttx.z = bexor;
						ttx.w = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
						*(uint4*)(&xs.hash[0]) = ttx;
					}
				}

				for (int k = 2; k != pos[i]; ++k)
				{
					u32 pindex = atomicAdd(&pairs_len, 1);
					if (pindex >= MAXPAIRS) break;
					u16 prev = ht[hr[i]][k];
					pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
				}
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		u32 pair = pairs[s];
		u32 i = __byte_perm(pair, 0, 0x4510);
		u32 k = __byte_perm(pair, 0, 0x4532);

		xors[2] = lastword2[i] ^ lastword2[k];
		if (xors[2] == 0)
			continue;

		*(uint2*)(&xors[0]) = lastword1[i] ^ lastword1[k];

		bexor = __byte_perm(xors[0], xors[1], 0x1076);
		xorbucketid = bexor >> (12 + RB);
		xorslot = atomicAdd(&eq->edata.nslots[6][xorbucketid], 1);
		if (xorslot >= NSLOTS) continue;
		slotsmall &xs = eq->treessmall[0][xorbucketid][xorslot];
		uint4 ttx;
		ttx.x = xors[1];
		ttx.y = xors[2];
		ttx.z = bexor;
		ttx.w = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
		*(uint4*)(&xs.hash[0]) = ttx;
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS>
__global__ void digit_7(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ u32 lastword[NSLOTS][2];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 bsize_sh;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	ht_len[threadid] = 0;
	if (threadid == (NRESTS - 1))
	{
		pairs_len = 0;
		next_pair = 0;
	}
	else if (threadid == (NRESTS - 33))
		bsize_sh = umin(eq->edata.nslots[6][bucketid], NSLOTS);

	slotsmall* buck = eq->treessmall[0][bucketid];

	u32 hr[3];
	int pos[3];
	pos[0] = pos[1] = pos[2] = SSM;

	u32 si[3];
	uint4 tt[3];

	__syncthreads();

	u32 bsize = bsize_sh;

#pragma unroll
	for (u32 i = 0; i != 3; ++i)
	{
		si[i] = i * NRESTS + threadid;
		if (si[i] >= bsize) break;

		const slotsmall* pslot1 = buck + si[i];

		// get xhash
		tt[i] = *(uint4*)(&pslot1->hash[0]);
		*(uint2*)(&lastword[si[i]][0]) = *(uint2*)(&tt[i].x);
		asm("bfe.u32 %0, %1, 12, %2;" : "=r"(hr[i]) : "r"(tt[i].z), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();

	u32 xors[2];
	u32 xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i != 3; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ *(uint2*)(&lastword[p][0]);

			if (xors[1] != 0)
			{
				asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(8 + RB), "r"(BUCKBITS));
				xorslot = atomicAdd(&eq->edata.nslots[7][xorbucketid], 1);
				if (xorslot < NSLOTS)
				{
					slotsmall &xs = eq->treessmall[1][xorbucketid][xorslot];
					uint4 ttx;
					ttx.x = xors[0];
					ttx.y = xors[1];
					ttx.z = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
					ttx.w = 0;
					*(uint4*)(&xs.hash[0]) = ttx;
				}
			}

			if (pos[i] > 1)
			{
				p = ht[hr[i]][1];

				*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ *(uint2*)(&lastword[p][0]);

				if (xors[1] != 0)
				{
					asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(8 + RB), "r"(BUCKBITS));
					xorslot = atomicAdd(&eq->edata.nslots[7][xorbucketid], 1);
					if (xorslot < NSLOTS)
					{
						slotsmall &xs = eq->treessmall[1][xorbucketid][xorslot];
						uint4 ttx;
						ttx.x = xors[0];
						ttx.y = xors[1];
						ttx.z = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
						ttx.w = 0;
						*(uint4*)(&xs.hash[0]) = ttx;
					}
				}

				for (int k = 2; k != pos[i]; ++k)
				{
					u32 pindex = atomicAdd(&pairs_len, 1);
					if (pindex >= MAXPAIRS) break;
					u16 prev = ht[hr[i]][k];
					pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
				}
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		int pair = pairs[s];
		u32 i = __byte_perm(pair, 0, 0x4510);
		u32 k = __byte_perm(pair, 0, 0x4532);

		*(uint2*)(&xors[0]) = *(uint2*)(&lastword[i][0]) ^ *(uint2*)(&lastword[k][0]);

		if (xors[1] == 0)
			continue;

		asm("bfe.u32 %0, %1, %2, %3;" : "=r"(xorbucketid) : "r"(xors[0]), "r"(8 + RB), "r"(BUCKBITS));
		xorslot = atomicAdd(&eq->edata.nslots[7][xorbucketid], 1);
		if (xorslot >= NSLOTS) continue;
		slotsmall &xs = eq->treessmall[1][xorbucketid][xorslot];
		uint4 tt;
		tt.x = xors[0];
		tt.y = xors[1];
		tt.z = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
		tt.w = 0;
		*(uint4*)(&xs.hash[0]) = tt;
	}
}


template <u32 RB, u32 SM, int SSM, typename PACKER, u32 MAXPAIRS>
__global__ void digit_8(equi<RB, SM>* eq)
{
	__shared__ u16 ht[NRESTS][(SSM - 1)];
	__shared__ u32 lastword[NSLOTS][2];
	__shared__ int ht_len[NRESTS];
	__shared__ int pairs[MAXPAIRS];
	__shared__ u32 pairs_len;
	__shared__ u32 bsize_sh;
	__shared__ u32 next_pair;

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
	ht_len[threadid] = 0;
	if (threadid == (NRESTS - 1))
	{
		next_pair = 0;
		pairs_len = 0;
	}
	else if (threadid == (NRESTS - 33))
		bsize_sh = umin(eq->edata.nslots[7][bucketid], NSLOTS);

	slotsmall* buck = eq->treessmall[1][bucketid];

	u32 hr[3];
	int pos[3];
	pos[0] = pos[1] = pos[2] = SSM;

	u32 si[3];
	uint2 tt[3];

	__syncthreads();

	u32 bsize = bsize_sh;

#pragma unroll
	for (u32 i = 0; i != 3; ++i)
	{
		si[i] = i * NRESTS + threadid;
		if (si[i] >= bsize) break;

		const slotsmall* pslot1 = buck + si[i];

		// get xhash
		tt[i] = *(uint2*)(&pslot1->hash[0]);
		*(uint2*)(&lastword[si[i]][0]) = *(uint2*)(&tt[i].x);
		asm("bfe.u32 %0, %1, 8, %2;" : "=r"(hr[i]) : "r"(tt[i].x), "r"(RB));
		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1)) ht[hr[i]][pos[i]] = si[i];
	}

	__syncthreads();

	u32 xors[2];
	u32 bexor, xorbucketid, xorslot;

#pragma unroll
	for (u32 i = 0; i != 3; ++i)
	{
		if (pos[i] >= SSM) continue;

		if (pos[i] > 0)
		{
			u16 p = ht[hr[i]][0];

			*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ *(uint2*)(&lastword[p][0]);

			if (xors[1] != 0)
			{
				bexor = __byte_perm(xors[0], xors[1], 0x0765);
				xorbucketid = bexor >> (12 + 8);
				xorslot = atomicAdd(&eq->edata.nslots8[xorbucketid], 1);
				if (xorslot < RB8_NSLOTS_LD)
				{
					slottiny &xs = eq->treestiny[0][xorbucketid][xorslot];
					uint2 tt;
					tt.x = xors[1];
					tt.y = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
					*(uint2*)(&xs.hash[0]) = tt;
				}
			}

			if (pos[i] > 1)
			{
				p = ht[hr[i]][1];

				*(uint2*)(&xors[0]) = *(uint2*)(&tt[i].x) ^ *(uint2*)(&lastword[p][0]);

				if (xors[1] != 0)
				{
					bexor = __byte_perm(xors[0], xors[1], 0x0765);
					xorbucketid = bexor >> (12 + 8);
					xorslot = atomicAdd(&eq->edata.nslots8[xorbucketid], 1);
					if (xorslot < RB8_NSLOTS_LD)
					{
						slottiny &xs = eq->treestiny[0][xorbucketid][xorslot];
						uint2 tt;
						tt.x = xors[1];
						tt.y = PACKER::set_bucketid_and_slots(bucketid, si[i], p, RB, SM);
						*(uint2*)(&xs.hash[0]) = tt;
					}
				}

				for (int k = 2; k != pos[i]; ++k)
				{
					u32 pindex = atomicAdd(&pairs_len, 1);
					if (pindex >= MAXPAIRS) break;
					u16 prev = ht[hr[i]][k];
					pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
				}
			}
		}
	}

	__syncthreads();

	// process pairs
	u32 plen = umin(pairs_len, MAXPAIRS);
	for (u32 s = atomicAdd(&next_pair, 1); s < plen; s = atomicAdd(&next_pair, 1))
	{
		int pair = pairs[s];
		u32 i = __byte_perm(pair, 0, 0x4510);
		u32 k = __byte_perm(pair, 0, 0x4532);

		*(uint2*)(&xors[0]) = *(uint2*)(&lastword[i][0]) ^ *(uint2*)(&lastword[k][0]);

		if (xors[1] == 0)
			continue;

		bexor = __byte_perm(xors[0], xors[1], 0x0765);
		xorbucketid = bexor >> (12 + 8);
		xorslot = atomicAdd(&eq->edata.nslots8[xorbucketid], 1);
		if (xorslot >= RB8_NSLOTS_LD) continue;
		slottiny &xs = eq->treestiny[0][xorbucketid][xorslot];
		uint2 tt;
		tt.x = xors[1];
		tt.y = PACKER::set_bucketid_and_slots(bucketid, i, k, RB, SM);
		*(uint2*)(&xs.hash[0]) = tt;
	}
}

/*
  Last round function is similar to previous ones but has different ending.
  We use warps to process final candidates. Each warp process one candidate.
  First two bidandsids (u32 of stored bucketid and two slotids) are retreived by
  lane 0 and lane 16, next four bidandsids by lane 0, 8, 16 and 24, ... until
  all lanes in warp have bidandsids from round 4. Next, each thread retreives
  16 indices. While doing so, indices are put into comparison using atomicExch
  to determine if there are duplicates (tromp's method). At the end, if no
  duplicates are found, candidate solution is saved (all indices). Note that this
  dup check method is not exact so CPU dup checking is needed after.
*/
template <u32 RB, u32 SM, int SSM, u32 FCT, typename PACKER, u32 MAXPAIRS, u32 DUPBITS, u32 W>
__global__ void digit_last_wdc(equi<RB, SM>* eq)
{
	__shared__ u8 shared_data[8192];
	int* ht_len = (int*)(&shared_data[0]);
	int* pairs = ht_len;
	u32* lastword = (u32*)(&shared_data[256 * 4]);
	u16* ht = (u16*)(&shared_data[256 * 4 + RB8_NSLOTS_LD * 4]);
	u32* pairs_len = (u32*)(&shared_data[8188]);

	const u32 threadid = threadIdx.x;
	const u32 bucketid = blockIdx.x;

	// reset hashtable len
#pragma unroll
	for (u32 i = 0; i != FCT; ++i)
		ht_len[(i * (256 / FCT)) + threadid] = 0;

	if (threadid == ((256 / FCT) - 1))
		*pairs_len = 0;

	slottiny* buck = eq->treestiny[0][bucketid];
	u32 bsize = umin(eq->edata.nslots8[bucketid], RB8_NSLOTS_LD);

	u32 si[3 * FCT];
	u32 hr[3 * FCT];
	int pos[3 * FCT];
	u32 lw[3 * FCT];
#pragma unroll
	for (u32 i = 0; i != (3 * FCT); ++i)
		pos[i] = SSM;

	__syncthreads();

#pragma unroll
	for (u32 i = 0; i != (3 * FCT); ++i)
	{
		si[i] = i * (256 / FCT) + threadid;
		if (si[i] >= bsize) break;

		const slottiny* pslot1 = buck + si[i];

		// get xhash
		uint2 tt = *(uint2*)(&pslot1->hash[0]);
		lw[i] = tt.x;
		lastword[si[i]] = lw[i];

		u32 a;
		asm("bfe.u32 %0, %1, 20, 8;" : "=r"(a) : "r"(lw[i]));
		hr[i] = a;

		pos[i] = atomicAdd(&ht_len[hr[i]], 1);
		if (pos[i] < (SSM - 1))
			ht[hr[i] * (SSM - 1) + pos[i]] = si[i];
	}

	__syncthreads();

#pragma unroll
	for (u32 i = 0; i != (3 * FCT); ++i)
	{
		if (pos[i] >= SSM) continue;

		for (int k = 0; k != pos[i]; ++k)
		{
			u16 prev = ht[hr[i] * (SSM - 1) + k];
			if (lw[i] != lastword[prev]) continue;
			u32 pindex = atomicAdd(pairs_len, 1);
			if (pindex >= MAXPAIRS) break;
			pairs[pindex] = __byte_perm(si[i], prev, 0x1054);
		}
	}

	__syncthreads();
	u32 plen = umin(*pairs_len, 64);

#define CALC_LEVEL(a, b, c, d) { \
	u32 plvl = levels[b]; \
	u32* bucks = eq->round4bidandsids[PACKER::get_bucketid(plvl, RB, SM)]; \
	u32 slot1 = PACKER::get_slot1(plvl, RB, SM); \
	u32 slot0 = PACKER::get_slot0(plvl, slot1, RB, SM); \
	levels[b] = bucks[slot1]; \
	levels[c] = bucks[slot0]; \
				}

#define CALC_LEVEL_SMALL(a, b, c, d) { \
	u32 plvl = levels[b]; \
	slotsmall* bucks = eq->treessmall[a][PACKER::get_bucketid(plvl, RB, SM)]; \
	u32 slot1 = PACKER::get_slot1(plvl, RB, SM); \
	u32 slot0 = PACKER::get_slot0(plvl, slot1, RB, SM); \
	levels[b] = bucks[slot1].hash[d]; \
	levels[c] = bucks[slot0].hash[d]; \
				}

	u32 lane = threadIdx.x & 0x1f;
	u32 par = threadIdx.x >> 5;

	u32* levels = (u32*)&pairs[MAXPAIRS + (par << DUPBITS)];
	u32* susp = levels;

	while (par < plen)
	{
		int pair = pairs[par];
		par += W;

		if (lane % 16 == 0)
		{
			u32 plvl;
			if (lane == 0) plvl = buck[__byte_perm(pair, 0, 0x4510)].hash[1];
			else plvl = buck[__byte_perm(pair, 0, 0x4532)].hash[1];
			slotsmall* bucks = eq->treessmall[1][PACKER::get_bucketid(plvl, RB, SM)];
			u32 slot1 = PACKER::get_slot1(plvl, RB, SM);
			u32 slot0 = PACKER::get_slot0(plvl, slot1, RB, SM);
			levels[lane] = bucks[slot1].hash[2];
			levels[lane + 8] = bucks[slot0].hash[2];
		}

		if (lane % 8 == 0)
			CALC_LEVEL_SMALL(0, lane, lane + 4, 3);

		if (lane % 4 == 0)
			CALC_LEVEL_SMALL(2, lane, lane + 2, 3);

		if (lane % 2 == 0)
			CALC_LEVEL(0, lane, lane + 1, 4);

		u32 ind[16];

		u32 f1 = levels[lane];
		const slottiny* buck_v4 = &eq->round3trees[PACKER::get_bucketid(f1, RB, SM)].treestiny[0];
		const u32 slot1_v4 = PACKER::get_slot1(f1, RB, SM);
		const u32 slot0_v4 = PACKER::get_slot0(f1, slot1_v4, RB, SM);

		susp[lane] = 0xffffffff;
		susp[32 + lane] = 0xffffffff;

#define CHECK_DUP(a) \
	__any(atomicExch(&susp[(ind[a] & ((1 << DUPBITS) - 1))], (ind[a] >> DUPBITS)) == (ind[a] >> DUPBITS))

		u32 f2 = buck_v4[slot1_v4].hash[1];
		const slottiny* buck_v3_1 = &eq->round2trees[PACKER::get_bucketid(f2, RB, SM)].treestiny[0];
		const u32 slot1_v3_1 = PACKER::get_slot1(f2, RB, SM);
		const u32 slot0_v3_1 = PACKER::get_slot0(f2, slot1_v3_1, RB, SM);

		susp[64 + lane] = 0xffffffff;
		susp[96 + lane] = 0xffffffff;

		u32 f0 = buck_v3_1[slot1_v3_1].hash[1];
		const slot* buck_v2_1 = eq->trees[0][PACKER::get_bucketid(f0, RB, SM)];
		const u32 slot1_v2_1 = PACKER::get_slot1(f0, RB, SM);
		const u32 slot0_v2_1 = PACKER::get_slot0(f0, slot1_v2_1, RB, SM);

		susp[128 + lane] = 0xffffffff;
		susp[160 + lane] = 0xffffffff;

		u32 f3 = buck_v2_1[slot1_v2_1].hash[6];
		const slot* buck_fin_1 = eq->round0trees[packer_default::get_bucketid(f3, 8, RB8_NSLOTS)];
		const u32 slot1_fin_1 = packer_default::get_slot1(f3, 8, RB8_NSLOTS);
		const u32 slot0_fin_1 = packer_default::get_slot0(f3, slot1_fin_1, 8, RB8_NSLOTS);

		susp[192 + lane] = 0xffffffff;
		susp[224 + lane] = 0xffffffff;

		ind[0] = buck_fin_1[slot1_fin_1].hash[7];
		if (CHECK_DUP(0)) continue;
		ind[1] = buck_fin_1[slot0_fin_1].hash[7];
		if (CHECK_DUP(1)) continue;

		u32 f4 = buck_v2_1[slot0_v2_1].hash[6];
		const slot* buck_fin_2 = eq->round0trees[packer_default::get_bucketid(f4, 8, RB8_NSLOTS)];
		const u32 slot1_fin_2 = packer_default::get_slot1(f4, 8, RB8_NSLOTS);
		const u32 slot0_fin_2 = packer_default::get_slot0(f4, slot1_fin_2, 8, RB8_NSLOTS);

		ind[2] = buck_fin_2[slot1_fin_2].hash[7];
		if (CHECK_DUP(2)) continue;
		ind[3] = buck_fin_2[slot0_fin_2].hash[7];
		if (CHECK_DUP(3)) continue;

		u32 f5 = buck_v3_1[slot0_v3_1].hash[1];
		const slot* buck_v2_2 = eq->trees[0][PACKER::get_bucketid(f5, RB, SM)];
		const u32 slot1_v2_2 = PACKER::get_slot1(f5, RB, SM);
		const u32 slot0_v2_2 = PACKER::get_slot0(f5, slot1_v2_2, RB, SM);

		u32 f6 = buck_v2_2[slot1_v2_2].hash[6];
		const slot* buck_fin_3 = eq->round0trees[packer_default::get_bucketid(f6, 8, RB8_NSLOTS)];
		const u32 slot1_fin_3 = packer_default::get_slot1(f6, 8, RB8_NSLOTS);
		const u32 slot0_fin_3 = packer_default::get_slot0(f6, slot1_fin_3, 8, RB8_NSLOTS);

		ind[4] = buck_fin_3[slot1_fin_3].hash[7];
		if (CHECK_DUP(4)) continue;
		ind[5] = buck_fin_3[slot0_fin_3].hash[7];
		if (CHECK_DUP(5)) continue;

		u32 f7 = buck_v2_2[slot0_v2_2].hash[6];
		const slot* buck_fin_4 = eq->round0trees[packer_default::get_bucketid(f7, 8, RB8_NSLOTS)];
		const u32 slot1_fin_4 = packer_default::get_slot1(f7, 8, RB8_NSLOTS);
		const u32 slot0_fin_4 = packer_default::get_slot0(f7, slot1_fin_4, 8, RB8_NSLOTS);

		ind[6] = buck_fin_4[slot1_fin_4].hash[7];
		if (CHECK_DUP(6)) continue;
		ind[7] = buck_fin_4[slot0_fin_4].hash[7];
		if (CHECK_DUP(7)) continue;

		u32 f8 = buck_v4[slot0_v4].hash[1];
		const slottiny* buck_v3_2 = &eq->round2trees[PACKER::get_bucketid(f8, RB, SM)].treestiny[0];
		const u32 slot1_v3_2 = PACKER::get_slot1(f8, RB, SM);
		const u32 slot0_v3_2 = PACKER::get_slot0(f8, slot1_v3_2, RB, SM);

		u32 f9 = buck_v3_2[slot1_v3_2].hash[1];
		const slot* buck_v2_3 = eq->trees[0][PACKER::get_bucketid(f9, RB, SM)];
		const u32 slot1_v2_3 = PACKER::get_slot1(f9, RB, SM);
		const u32 slot0_v2_3 = PACKER::get_slot0(f9, slot1_v2_3, RB, SM);

		u32 f10 = buck_v2_3[slot1_v2_3].hash[6];
		const slot* buck_fin_5 = eq->round0trees[packer_default::get_bucketid(f10, 8, RB8_NSLOTS)];
		const u32 slot1_fin_5 = packer_default::get_slot1(f10, 8, RB8_NSLOTS);
		const u32 slot0_fin_5 = packer_default::get_slot0(f10, slot1_fin_5, 8, RB8_NSLOTS);

		ind[8] = buck_fin_5[slot1_fin_5].hash[7];
		if (CHECK_DUP(8)) continue;
		ind[9] = buck_fin_5[slot0_fin_5].hash[7];
		if (CHECK_DUP(9)) continue;

		u32 f11 = buck_v2_3[slot0_v2_3].hash[6];
		const slot* buck_fin_6 = eq->round0trees[packer_default::get_bucketid(f11, 8, RB8_NSLOTS)];
		const u32 slot1_fin_6 = packer_default::get_slot1(f11, 8, RB8_NSLOTS);
		const u32 slot0_fin_6 = packer_default::get_slot0(f11, slot1_fin_6, 8, RB8_NSLOTS);

		ind[10] = buck_fin_6[slot1_fin_6].hash[7];
		if (CHECK_DUP(10)) continue;
		ind[11] = buck_fin_6[slot0_fin_6].hash[7];
		if (CHECK_DUP(11)) continue;

		u32 f12 = buck_v3_2[slot0_v3_2].hash[1];
		const slot* buck_v2_4 = eq->trees[0][PACKER::get_bucketid(f12, RB, SM)];
		const u32 slot1_v2_4 = PACKER::get_slot1(f12, RB, SM);
		const u32 slot0_v2_4 = PACKER::get_slot0(f12, slot1_v2_4, RB, SM);

		u32 f13 = buck_v2_4[slot1_v2_4].hash[6];
		const slot* buck_fin_7 = eq->round0trees[packer_default::get_bucketid(f13, 8, RB8_NSLOTS)];
		const u32 slot1_fin_7 = packer_default::get_slot1(f13, 8, RB8_NSLOTS);
		const u32 slot0_fin_7 = packer_default::get_slot0(f13, slot1_fin_7, 8, RB8_NSLOTS);

		ind[12] = buck_fin_7[slot1_fin_7].hash[7];
		if (CHECK_DUP(12)) continue;
		ind[13] = buck_fin_7[slot0_fin_7].hash[7];
		if (CHECK_DUP(13)) continue;

		u32 f14 = buck_v2_4[slot0_v2_4].hash[6];
		const slot* buck_fin_8 = eq->round0trees[packer_default::get_bucketid(f14, 8, RB8_NSLOTS)];
		const u32 slot1_fin_8 = packer_default::get_slot1(f14, 8, RB8_NSLOTS);
		const u32 slot0_fin_8 = packer_default::get_slot0(f14, slot1_fin_8, 8, RB8_NSLOTS);

		ind[14] = buck_fin_8[slot1_fin_8].hash[7];
		if (CHECK_DUP(14)) continue;
		ind[15] = buck_fin_8[slot0_fin_8].hash[7];
		if (CHECK_DUP(15)) continue;

		u32 soli;
		if (lane == 0)
		{
			soli = atomicAdd(&eq->edata.srealcont.nsols, 1);
		}
		soli = __shfl(soli, 0);

		if (soli < MAXREALSOLS)
		{
			u32 pos = lane << 4;
			*(uint4*)(&eq->edata.srealcont.sols[soli][pos]) = *(uint4*)(&ind[0]);
			*(uint4*)(&eq->edata.srealcont.sols[soli][pos + 4]) = *(uint4*)(&ind[4]);
			*(uint4*)(&eq->edata.srealcont.sols[soli][pos + 8]) = *(uint4*)(&ind[8]);
			*(uint4*)(&eq->edata.srealcont.sols[soli][pos + 12]) = *(uint4*)(&ind[12]);
		}
	}
}


std::mutex dev_init;
int dev_init_done[8] = { 0 };


__host__ int compu32(const void *pa, const void *pb)
{
	uint32_t a = *(uint32_t *)pa, b = *(uint32_t *)pb;
	return a<b ? -1 : a == b ? 0 : +1;
}


__host__ bool duped(uint32_t* prf)
{
	uint32_t sortprf[512];
	memcpy(sortprf, prf, sizeof(uint32_t) * 512);
	qsort(sortprf, 512, sizeof(uint32_t), &compu32);
	for (uint32_t i = 1; i<512; i++)
		if (sortprf[i] <= sortprf[i - 1])
			return true;
	return false;
}


__host__ void sort_pair(uint32_t *a, uint32_t len)
{
	uint32_t    *b = a + len;
	uint32_t     tmp, need_sorting = 0;
	for (uint32_t i = 0; i < len; i++)
		if (need_sorting || a[i] > b[i])
		{
			need_sorting = 1;
			tmp = a[i];
			a[i] = b[i];
			b[i] = tmp;
		}
		else if (a[i] < b[i])
			return;
}


__host__ void setheader(blake2b_state *ctx, const char *header, const u32 headerLen, const char* nce, const u32 nonceLen)
{
	uint32_t le_N = WN;
	uint32_t le_K = WK;
	uchar personal[] = "ZcashPoW01230123";
	memcpy(personal + 8, &le_N, 4);
	memcpy(personal + 12, &le_K, 4);
	blake2b_param P[1];
	P->digest_length = HASHOUT;
	P->key_length = 0;
	P->fanout = 1;
	P->depth = 1;
	P->leaf_length = 0;
	P->node_offset = 0;
	P->node_depth = 0;
	P->inner_length = 0;
	memset(P->reserved, 0, sizeof(P->reserved));
	memset(P->salt, 0, sizeof(P->salt));
	memcpy(P->personal, (const uint8_t *)personal, 16);
	blake2b_init_param(ctx, P);
	blake2b_update(ctx, (const uchar *)header, headerLen);
	blake2b_update(ctx, (const uchar *)nce, nonceLen);
}


#ifdef WIN32
typedef hipError_t(CUDAAPI *dec_cuDeviceGet)(hipDevice_t*, int);
typedef hipError_t(CUDAAPI *dec_cuCtxCreate)(hipCtx_t*, unsigned int, hipDevice_t);
typedef hipError_t(CUDAAPI *dec_cuCtxPushCurrent)(hipCtx_t);
typedef hipError_t(CUDAAPI *dec_cuCtxDestroy)(hipCtx_t);

dec_cuDeviceGet _cuDeviceGet = nullptr;
dec_cuCtxCreate _cuCtxCreate = nullptr;
dec_cuCtxPushCurrent _cuCtxPushCurrent = nullptr;
dec_cuCtxDestroy _cuCtxDestroy = nullptr;
#endif


template <u32 RB, u32 SM, u32 SSM, u32 THREADS, typename PACKER>
__host__ eq_cuda_context<RB, SM, SSM, THREADS, PACKER>::eq_cuda_context(int id)
	: device_id(id)
{
	solutions = nullptr;

	dev_init.lock();
	if (!dev_init_done[device_id])
	{
		// only first thread shall init device
		checkCudaErrors(hipSetDevice(device_id));
		checkCudaErrors(hipDeviceReset());
		checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

		pctx = nullptr;
	}
	else
	{
		// create new context
		hipDevice_t dev;

#ifdef WIN32
		if (_cuDeviceGet == nullptr)
		{
			HMODULE hmod = LoadLibraryA("nvcuda.dll");
			if (hmod == NULL)
				throw std::runtime_error("Failed to load nvcuda.dll");
			_cuDeviceGet = (dec_cuDeviceGet)GetProcAddress(hmod, "hipDeviceGet");
			if (_cuDeviceGet == nullptr)
				throw std::runtime_error("Failed to get hipDeviceGet address");
			_cuCtxCreate = (dec_cuCtxCreate)GetProcAddress(hmod, "hipCtxCreate");
			if (_cuCtxCreate == nullptr)
				throw std::runtime_error("Failed to get hipCtxCreate address");
			_cuCtxPushCurrent = (dec_cuCtxPushCurrent)GetProcAddress(hmod, "hipCtxPushCurrent");
			if (_cuCtxPushCurrent == nullptr)
				throw std::runtime_error("Failed to get hipCtxPushCurrent address");
			_cuCtxDestroy = (dec_cuCtxDestroy)GetProcAddress(hmod, "hipCtxDestroy");
			if (_cuCtxDestroy == nullptr)
				throw std::runtime_error("Failed to get hipCtxDestroy address");
		}


		checkCudaDriverErrors(_cuDeviceGet(&dev, device_id));
		checkCudaDriverErrors(_cuCtxCreate(&pctx, hipDeviceScheduleBlockingSync, dev));
		checkCudaDriverErrors(_cuCtxPushCurrent(pctx));
#else
		checkCudaDriverErrors(hipDeviceGet(&dev, device_id));
		checkCudaDriverErrors(hipCtxCreate(&pctx, hipDeviceScheduleBlockingSync, dev));
		checkCudaDriverErrors(hipCtxPushCurrent(pctx));
#endif
	}
	++dev_init_done[device_id];
	dev_init.unlock();

	if (hipMalloc((void**)&device_eq, sizeof(equi<RB, SM>)) != hipSuccess)
		throw std::runtime_error("CUDA: failed to alloc memory");

	solutions = (scontainerreal*)malloc(sizeof(scontainerreal));
}


template <u32 RB, u32 SM, u32 SSM, u32 THREADS, typename PACKER>
__host__ void eq_cuda_context<RB, SM, SSM, THREADS, PACKER>::solve(const char *tequihash_header,
	unsigned int tequihash_header_len,
	const char* nonce,
	unsigned int nonce_len,
	std::function<bool()> cancelf,
	std::function<void(const std::vector<uint32_t>&, size_t, const unsigned char*)> solutionf,
	std::function<void(void)> hashdonef)
{
	blake2b_state blake_ctx;

	int blocks = NBUCKETS;

	setheader(&blake_ctx, tequihash_header, tequihash_header_len, nonce, nonce_len);

	// todo: improve
	// djezo solver allows last 4 bytes of nonce to be iterrated
	// this can be used to create internal loop - calc initial blake hash only once, then load 8*8 bytes on device (blake state h)
	// then just iterate nn++
	// less CPU load, 1 hipMemcpy less -> faster
	//u32 nn = *(u32*)&nonce[28];
	u32 nn = 0;

	checkCudaErrors(hipMemcpy(&device_eq->blake_h, &blake_ctx.h, sizeof(u64) * 8, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemset(&device_eq->edata, 0, sizeof(device_eq->edata)));

	digit_first<RB, SM, PACKER> << <NBLOCKS / FD_THREADS, FD_THREADS >> >(device_eq, nn);

	digit_1<RB, SM, SSM, PACKER, 4 * NRESTS, 512> << <4096, 512 >> >(device_eq);

	digit_2<RB, SM, SSM, PACKER, 4 * NRESTS, THREADS> << <blocks, THREADS >> >(device_eq);

	digit_3<RB, SM, SSM, PACKER, 4 * NRESTS, THREADS> << <blocks, THREADS >> >(device_eq);

	if (cancelf()) return;

	digit_4<RB, SM, SSM, PACKER, 4 * NRESTS, THREADS> << <blocks, THREADS >> >(device_eq);

	digit_5<RB, SM, SSM, PACKER, 4 * NRESTS, THREADS> << <blocks, THREADS >> >(device_eq);

	digit_6<RB, SM, SSM - 1, PACKER, 3 * NRESTS> << <blocks, NRESTS >> >(device_eq);

	digit_7<RB, SM, SSM - 1, PACKER, 3 * NRESTS> << <blocks, NRESTS >> >(device_eq);

	digit_8<RB, SM, SSM - 1, PACKER, 3 * NRESTS> << <blocks, NRESTS >> >(device_eq);

	digit_last_wdc<RB, SM, SSM - 3, 2, PACKER, 64, 8, 4> << <4096, 256 / 2 >> >(device_eq);

	checkCudaErrors(hipMemcpy(solutions, &device_eq->edata.srealcont, (MAXREALSOLS * (512 * 4)) + 4, hipMemcpyDeviceToHost));

	//printf("nsols: %u\n", solutions->nsols);
	//if (solutions->nsols > 9)
	//	printf("missing sol, total: %u\n", solutions->nsols);

	for (u32 s = 0; (s < solutions->nsols) && (s < MAXREALSOLS); s++)
	{
		// remove dups on CPU (dup removal on GPU is not fully exact and can pass on some invalid solutions)
		if (duped(solutions->sols[s])) continue;

		// perform sort of pairs
		for (uint32_t level = 0; level < 9; level++)
			for (uint32_t i = 0; i < (1 << 9); i += (2 << level))
				sort_pair(&solutions->sols[s][i], 1 << level);

		std::vector<uint32_t> index_vector(PROOFSIZE);
		for (u32 i = 0; i < PROOFSIZE; i++) {
			index_vector[i] = solutions->sols[s][i];
		}
		
		solutionf(index_vector, DIGITBITS, nullptr);
	}

	hashdonef();
}


template <u32 RB, u32 SM, u32 SSM, u32 THREADS, typename PACKER>
__host__ eq_cuda_context<RB, SM, SSM, THREADS, PACKER>::~eq_cuda_context()
{
	if (solutions)
		free(solutions);

	hipFree(device_eq);

	if (pctx)
	{
		// non primary thread, destroy context
#ifdef WIN32
		checkCudaDriverErrors(_cuCtxDestroy(pctx));
#else
		checkCudaDriverErrors(hipCtxDestroy(pctx));
#endif
	}
	else
	{
		checkCudaErrors(hipDeviceReset());

		dev_init_done[device_id] = 0;
	}
}


#ifdef CONFIG_MODE_1
template class eq_cuda_context<CONFIG_MODE_1>;
#endif

#ifdef CONFIG_MODE_2
template class eq_cuda_context<CONFIG_MODE_2>;
#endif

#ifdef CONFIG_MODE_3
template class eq_cuda_context<CONFIG_MODE_3>;
#endif
