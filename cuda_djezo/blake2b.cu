#include "hip/hip_runtime.h"
// Blake2-B CUDA Implementation
// tpruvot@github July 2016
// permission granted to use under MIT license
// modified for use in Zcash by John Tromp September 2016

/**
 * uint2 direct ops by c++ operator definitions
 */
static __device__ __forceinline__ uint2 operator^ (uint2 a, uint2 b) {
  return make_uint2(a.x ^ b.x, a.y ^ b.y);
}
static __forceinline__ __device__ uint4 operator^ (uint4 a, uint4 b) { 
	return make_uint4(a.x ^ b.x, a.y ^ b.y, a.z ^ b.z, a.w ^ b.w); 
}
// uint2 ROR/ROL methods
__device__ __forceinline__ uint2 ROR2(const uint2 a, const int offset) {
  uint2 result;
#if __CUDA_ARCH__ > 300
/*  if (offset < 32) {
          asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
          asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
  } else *//* if (offset < 64) */ {
          /* offset SHOULD BE < 64 ! */
          asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
          asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
  }
#else
  if (!offset)
          result = a;
  else if (offset < 32) {
          result.y = ((a.y >> offset) | (a.x << (32 - offset)));
          result.x = ((a.x >> offset) | (a.y << (32 - offset)));
  } else if (offset == 32) {
          result.y = a.x;
          result.x = a.y;
  } else {
          result.y = ((a.x >> (offset - 32)) | (a.y << (64 - offset)));
          result.x = ((a.y >> (offset - 32)) | (a.x << (64 - offset)));
  }
#endif
  return result;
}
__device__ __forceinline__ uint2 SWAPUINT2(uint2 value) {
  return make_uint2(value.y, value.x);
}
#ifdef __CUDA_ARCH__
__device__ __inline__ uint2 ROR24(const uint2 a) {
  uint2 result;
  result.x = __byte_perm(a.y, a.x, 0x2107);
  result.y = __byte_perm(a.y, a.x, 0x6543);
  return result;
}
__device__ __inline__ uint2 ROR16(const uint2 a) {
  uint2 result;
  result.x = __byte_perm(a.y, a.x, 0x1076);
  result.y = __byte_perm(a.y, a.x, 0x5432);
  return result;
}
#else
#define ROR24(u) ROR2(u,24)
#define ROR16(u) ROR2(u,16)
#endif

typedef uint64_t u64;

static __constant__ const int8_t blake2b_sigma[12][16] = {
  { 0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15 } ,
  { 14, 10, 4,  8,  9,  15, 13, 6,  1,  12, 0,  2,  11, 7,  5,  3  } ,
  { 11, 8,  12, 0,  5,  2,  15, 13, 10, 14, 3,  6,  7,  1,  9,  4  } ,
  { 7,  9,  3,  1,  13, 12, 11, 14, 2,  6,  5,  10, 4,  0,  15, 8  } ,
  { 9,  0,  5,  7,  2,  4,  10, 15, 14, 1,  11, 12, 6,  8,  3,  13 } ,
  { 2,  12, 6,  10, 0,  11, 8,  3,  4,  13, 7,  5,  15, 14, 1,  9  } ,
  { 12, 5,  1,  15, 14, 13, 4,  10, 0,  7,  6,  3,  9,  2,  8,  11 } ,
  { 13, 11, 7,  14, 12, 1,  3,  9,  5,  0,  15, 4,  8,  6,  2,  10 } ,
  { 6,  15, 14, 9,  11, 3,  0,  8,  12, 2,  13, 7,  1,  4,  10, 5  } ,
  { 10, 2,  8,  4,  7,  6,  1,  5,  15, 11, 9,  14, 3,  12, 13, 0  } ,
  { 0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15 } ,
  { 14, 10, 4,  8,  9,  15, 13, 6,  1,  12, 0,  2,  11, 7,  5,  3  }
};

__device__ __constant__
static const u64 blake_iv[] = 
{
	0x6a09e667f3bcc908, 0xbb67ae8584caa73b,
	0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1,
	0x510e527fade682d1, 0x9b05688c2b3e6c1f,
	0x1f83d9abfb41bd6b, 0x5be0cd19137e2179,
};

__device__ __forceinline__
static void G(const int r, const int i, u64 &a, u64 &b, u64 &c, u64 &d, u64 const m[16]) {
  a = a + b + m[ blake2b_sigma[r][2*i] ];
  ((uint2*)&d)[0] = SWAPUINT2( ((uint2*)&d)[0] ^ ((uint2*)&a)[0] );
  c = c + d;
  ((uint2*)&b)[0] = ROR24( ((uint2*)&b)[0] ^ ((uint2*)&c)[0] );
  a = a + b + m[ blake2b_sigma[r][2*i+1] ];
  ((uint2*)&d)[0] = ROR16( ((uint2*)&d)[0] ^ ((uint2*)&a)[0] );
  c = c + d;
  ((uint2*)&b)[0] = ROR2( ((uint2*)&b)[0] ^ ((uint2*)&c)[0], 63U);
}

//__device__ __forceinline__
//static void G2(u64 &a, u64 &b, u64 &c, u64 &d, u64 x, u64 y) {
//	a = a + b + x;
//	((uint2*)&d)[0] = SWAPUINT2(((uint2*)&d)[0] ^ ((uint2*)&a)[0]);
//	c = c + d;
//	((uint2*)&b)[0] = ROR24(((uint2*)&b)[0] ^ ((uint2*)&c)[0]);
//	a = a + b + y;
//	((uint2*)&d)[0] = ROR16(((uint2*)&d)[0] ^ ((uint2*)&a)[0]);
//	c = c + d;
//	((uint2*)&b)[0] = ROR2(((uint2*)&b)[0] ^ ((uint2*)&c)[0], 63U);
//}

__device__ __forceinline__
static void G2(u64 & a, u64 & b, u64 & c, u64 & d, u64 x, u64 y) {
	a = a + b + x;
	((uint2*)&d)[0] = SWAPUINT2(((uint2*)&d)[0] ^ ((uint2*)&a)[0]);
	c = c + d;
	((uint2*)&b)[0] = ROR24(((uint2*)&b)[0] ^ ((uint2*)&c)[0]);
	a = a + b + y;
	((uint2*)&d)[0] = ROR16(((uint2*)&d)[0] ^ ((uint2*)&a)[0]);
	c = c + d;
	((uint2*)&b)[0] = ROR2(((uint2*)&b)[0] ^ ((uint2*)&c)[0], 63U);
}

#define ROUND(r) \
  G(r, 0, v[0], v[4], v[ 8], v[12], m); \
  G(r, 1, v[1], v[5], v[ 9], v[13], m); \
  G(r, 2, v[2], v[6], v[10], v[14], m); \
  G(r, 3, v[3], v[7], v[11], v[15], m); \
  G(r, 4, v[0], v[5], v[10], v[15], m); \
  G(r, 5, v[1], v[6], v[11], v[12], m); \
  G(r, 6, v[2], v[7], v[ 8], v[13], m); \
  G(r, 7, v[3], v[4], v[ 9], v[14], m);


__forceinline__ __device__ void blake2b_gpu_hash3(uint64_t* h, u32 idx, u32 nonce) {
	u64 m = (u64)idx << 32 | (u64)nonce;

	u64 v[16];

	v[0] = h[0];
	v[1] = h[1];
	v[2] = h[2];
	v[3] = h[3];
	v[4] = h[4];
	v[5] = h[5];
	v[6] = h[6];
	v[7] = h[7];
	v[8] = blake_iv[0];
	v[9] = blake_iv[1];
	v[10] = blake_iv[2];
	v[11] = blake_iv[3];
	v[12] = blake_iv[4] ^ (128 + 16);
	v[13] = blake_iv[5];
	v[14] = blake_iv[6] ^ 0xffffffffffffffff;
	v[15] = blake_iv[7];

	// mix 1
	G2(v[0], v[4], v[8], v[12], 0, m);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 2
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], m, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 3
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, m);
	G2(v[3], v[4], v[9], v[14], 0, 0);
	
	// mix 4
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, m);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 5
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, m);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 6
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], m, 0);

	// mix 7
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], m, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 8
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, m);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 9
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], m, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 10
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], m, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 11
	G2(v[0], v[4], v[8], v[12], 0, m);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], 0, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	// mix 12
	G2(v[0], v[4], v[8], v[12], 0, 0);
	G2(v[1], v[5], v[9], v[13], 0, 0);
	G2(v[2], v[6], v[10], v[14], 0, 0);
	G2(v[3], v[7], v[11], v[15], 0, 0);
	G2(v[0], v[5], v[10], v[15], m, 0);
	G2(v[1], v[6], v[11], v[12], 0, 0);
	G2(v[2], v[7], v[8], v[13], 0, 0);
	G2(v[3], v[4], v[9], v[14], 0, 0);

	h[0] ^= v[0] ^ v[8];
	h[1] ^= v[1] ^ v[9];
	h[2] ^= v[2] ^ v[10];
	h[3] ^= v[3] ^ v[11];
	h[4] ^= v[4] ^ v[12];
	h[5] ^= v[5] ^ v[13];
	h[6] ^= v[6] ^ v[14];
}


__forceinline__ __device__ void blake2b_gpu_hash2(uint64_t* h, u32 idx) {
	u64 m[16] = { 0 };
	u32* ptr = (u32*)&m[1];

	ptr[1] = idx;

	u64 v[16];

	v[0] = h[0];
	v[1] = h[1];
	v[2] = h[2];
	v[3] = h[3];
	v[4] = h[4];
	v[5] = h[5];
	v[6] = h[6];
	v[7] = h[7];
	v[8] = 0x6a09e667f3bcc908;
	v[9] = 0xbb67ae8584caa73b;
	v[10] = 0x3c6ef372fe94f82b;
	v[11] = 0xa54ff53a5f1d36f1;
	v[12] = 0x510e527fade682d1 ^ (128 + 16);
	v[13] = 0x9b05688c2b3e6c1f;
	v[14] = 0x1f83d9abfb41bd6b ^ 0xffffffffffffffff;
	v[15] = 0x5be0cd19137e2179;

	ROUND(0);
	ROUND(1);
	ROUND(2);
	ROUND(3);
	ROUND(4);
	ROUND(5);
	ROUND(6);
	ROUND(7);
	ROUND(8);
	ROUND(9);
	ROUND(10);
	ROUND(11);

	h[0] ^= v[0] ^ v[8];
	h[1] ^= v[1] ^ v[9];
	h[2] ^= v[2] ^ v[10];
	h[3] ^= v[3] ^ v[11];
	h[4] ^= v[4] ^ v[12];
	h[5] ^= v[5] ^ v[13];
	h[6] ^= v[6] ^ v[14];
	//h[7] ^= v[7] ^ v[15];
	//memcpy(hash, (uchar *)h, outlen);
}