#include "hip/hip_runtime.h"
// Equihash CUDA solver
// Copyright (c) 2016 John Tromp

#define XINTREE
#define UNROLL
#define htole32(x) (x)
#define HAVE_DECL_HTOLE32 1

#include "../cpu_tromp/equi.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <functional>
#include <vector>
#include <iostream>

#include "eqcuda.hpp"

#include "blake2b.cu"


typedef uint16_t u16;
typedef uint64_t u64;


#ifndef RESTBITS
#define RESTBITS	4
#endif

// 2_log of number of buckets
#define BUCKBITS (DIGITBITS-RESTBITS)

#ifndef SAVEMEM
#if RESTBITS == 4
// can't save memory in such small buckets
#define SAVEMEM 1
#elif RESTBITS >= 8
// take advantage of law of large numbers (sum of 2^8 random numbers)
// this reduces (200,9) memory to under 144MB, with negligible discarding
#define SAVEMEM 9/14
#endif
#endif

// number of buckets
static const u32 NBUCKETS = 1 << BUCKBITS;
// bucket mask
static const u32 BUCKMASK = NBUCKETS - 1;
// 2_log of number of slots per bucket
static const u32 SLOTBITS = RESTBITS + 1 + 1;
static const u32 SLOTRANGE = 1 << SLOTBITS;
// number of slots per bucket
static const u32 NSLOTS = SLOTRANGE * SAVEMEM;
// SLOTBITS mask
static const u32 SLOTMASK = SLOTRANGE - 1;
// number of possible values of xhash (rest of n) bits
static const u32 NRESTS = 1 << RESTBITS;
// RESTBITS mask
static const u32 RESTMASK = NRESTS - 1;
// number of blocks of hashes extracted from single 512 bit blake2b output
static const u32 NBLOCKS = (NHASHES + HASHESPERBLAKE - 1) / HASHESPERBLAKE;
// nothing larger found in 100000 runs
static const u32 MAXSOLS = 8;

// tree node identifying its children as two different slots in
// a bucket on previous layer with the same rest bits (x-tra hash)
struct tree {
	u32 bid_s0_s1_x; // manual bitfields

	__device__ tree(const u32 idx, const u32 xh) {
		bid_s0_s1_x = idx << RESTBITS | xh;
	}
	__device__ tree(const u32 idx) {
		bid_s0_s1_x = idx;
	}
	__device__ tree(const u32 bid, const u32 s0, const u32 s1, const u32 xh) {
#ifdef XINTREE
		bid_s0_s1_x = ((((bid << SLOTBITS) | s0) << SLOTBITS) | s1) << RESTBITS | xh;
#else
		bid_s0_s1_x = (((bid << SLOTBITS) | s0) << SLOTBITS) | s1;
#endif
	}
	__device__ u32 getindex() const {
#ifdef XINTREE
		return bid_s0_s1_x >> RESTBITS;
#else
		return bid_s0_s1_x;
#endif
	}
	__device__ u32 bucketid() const {
#ifdef XINTREE
		return bid_s0_s1_x >> (2 * SLOTBITS + RESTBITS);
#else
		return bid_s0_s1_x >> (2 * SLOTBITS);
#endif
	}
	__device__ u32 slotid0() const {
#ifdef XINTREE
		return (bid_s0_s1_x >> SLOTBITS + RESTBITS) & SLOTMASK;
#else
		return (bid_s0_s1_x >> SLOTBITS) & SLOTMASK;
#endif
	}
	__device__ u32 slotid1() const {
#ifdef XINTREE
		return (bid_s0_s1_x >> RESTBITS) & SLOTMASK;
#else
		return bid_s0_s1_x & SLOTMASK;
#endif
	}
	__device__ u32 xhash() const {
		return bid_s0_s1_x & RESTMASK;
	}
};

union hashunit {
	u32 word;
	uchar bytes[sizeof(u32)];
};

#define WORDS(bits)     ((bits + 31) / 32)
#define HASHWORDS0 WORDS(WN - DIGITBITS + RESTBITS)
#define HASHWORDS1 WORDS(WN - 2*DIGITBITS + RESTBITS)

struct slot0 {
	tree attr;
	hashunit hash[HASHWORDS0];
};

struct slot1 {
	tree attr;
	hashunit hash[HASHWORDS1];
};

// a bucket is NSLOTS treenodes
typedef slot0 bucket0[NSLOTS];
typedef slot1 bucket1[NSLOTS];
// the N-bit hash consists of K+1 n-bit "digits"
// each of which corresponds to a layer of NBUCKETS buckets
typedef bucket0 digit0[NBUCKETS];
typedef bucket1 digit1[NBUCKETS];

// size (in bytes) of hash in round 0 <= r < WK
u32 hhashsize(const u32 r) {
#ifdef XINTREE
	const u32 hashbits = WN - (r + 1) * DIGITBITS;
#else
	const u32 hashbits = WN - (r + 1) * DIGITBITS + RESTBITS;
#endif
	return (hashbits + 7) / 8;
}
// size (in bytes) of hash in round 0 <= r < WK
__device__ u32 hashsize(const u32 r) {
#ifdef XINTREE
	const u32 hashbits = WN - (r + 1) * DIGITBITS;
#else
	const u32 hashbits = WN - (r + 1) * DIGITBITS + RESTBITS;
#endif
	return (hashbits + 7) / 8;
}

u32 hhashwords(u32 bytes) {
	return (bytes + 3) / 4;
}

__device__ u32 hashwords(u32 bytes) {
	return (bytes + 3) / 4;
}

// manages hash and tree data
struct htalloc {
	bucket0 *trees0[(WK + 1) / 2];
	bucket1 *trees1[WK / 2];
};

typedef u32 bsizes[NBUCKETS];

struct __align__(64) equi {
	blake2b_state blake_ctx;
	htalloc hta;
	bsizes *nslots;
	proof *sols;
	u32 nsols;
	u32 nthreads;
	equi(const u32 n_threads) {
		nthreads = n_threads;
	}
	void* operator new(size_t i)
	{
		return _mm_malloc(i, 64);
	}
	void operator delete(void* p) {
		_mm_free(p);
	}
	void setheadernonce(const char *header, const u32 len, const char* nonce, const u32 nlen) {
		setheader(&blake_ctx, header, len, nonce, nlen);
		checkCudaErrors(hipMemset(nslots, 0, NBUCKETS * sizeof(u32)));
		nsols = 0;
	}
	__device__ u32 getnslots0(const u32 bid) {
		u32 &nslot = nslots[0][bid];
		const u32 n = min(nslot, NSLOTS);
		nslot = 0;
		return n;
	}
	__device__ u32 getnslots1(const u32 bid) {
		u32 &nslot = nslots[1][bid];
		const u32 n = min(nslot, NSLOTS);
		nslot = 0;
		return n;
	}
	__device__ void orderindices(u32 *indices, u32 size) {
		if (indices[0] > indices[size]) {
			for (u32 i = 0; i < size; i++) {
				const u32 tmp = indices[i];
				indices[i] = indices[size + i];
				indices[size + i] = tmp;
			}
		}
	}
	__device__ void listindices1(const tree t, u32 *indices) {
		const bucket0 &buck = hta.trees0[0][t.bucketid()];
		const u32 size = 1 << 0;
		indices[0] = buck[t.slotid0()].attr.getindex();
		indices[size] = buck[t.slotid1()].attr.getindex();
		orderindices(indices, size);
	}
	__device__ void listindices2(const tree t, u32 *indices) {
		const bucket1 &buck = hta.trees1[0][t.bucketid()];
		const u32 size = 1 << 1;
		listindices1(buck[t.slotid0()].attr, indices);
		listindices1(buck[t.slotid1()].attr, indices + size);
		orderindices(indices, size);
	}
	__device__ void listindices3(const tree t, u32 *indices) {
		const bucket0 &buck = hta.trees0[1][t.bucketid()];
		const u32 size = 1 << 2;
		listindices2(buck[t.slotid0()].attr, indices);
		listindices2(buck[t.slotid1()].attr, indices + size);
		orderindices(indices, size);
	}
	__device__ void listindices4(const tree t, u32 *indices) {
		const bucket1 &buck = hta.trees1[1][t.bucketid()];
		const u32 size = 1 << 3;
		listindices3(buck[t.slotid0()].attr, indices);
		listindices3(buck[t.slotid1()].attr, indices + size);
		orderindices(indices, size);
	}
	__device__ void listindices5(const tree t, u32 *indices) {
		const bucket0 &buck = hta.trees0[2][t.bucketid()];
		const u32 size = 1 << 4;
		listindices4(buck[t.slotid0()].attr, indices);
		listindices4(buck[t.slotid1()].attr, indices+size);
		orderindices(indices, size);
	}
	__device__ void listindices6(const tree t, u32 *indices) {
		const bucket1 &buck = hta.trees1[2][t.bucketid()];
		const u32 size = 1 << 5;
		listindices5(buck[t.slotid0()].attr, indices);
		listindices5(buck[t.slotid1()].attr, indices+size);
		orderindices(indices, size);
	}
	__device__ void listindices7(const tree t, u32 *indices) {
		const bucket0 &buck = hta.trees0[3][t.bucketid()];
		const u32 size = 1 << 6;
		listindices6(buck[t.slotid0()].attr, indices);
		listindices6(buck[t.slotid1()].attr, indices+size);
		orderindices(indices, size);
	}
	__device__ void listindices8(const tree t, u32 *indices) {
		const bucket1 &buck = hta.trees1[3][t.bucketid()];
		const u32 size = 1 << 7;
		listindices7(buck[t.slotid0()].attr, indices);
		listindices7(buck[t.slotid1()].attr, indices+size);
		orderindices(indices, size);
	}
	__device__ void listindices9(const tree t, u32 *indices) {
		const bucket0 &buck = hta.trees0[4][t.bucketid()];
		const u32 size = 1 << 8;
		listindices8(buck[t.slotid0()].attr, indices);
		listindices8(buck[t.slotid1()].attr, indices+size);
		orderindices(indices, size);
	}
	__device__ void candidate(const tree t) {
		proof prf;
#if WK==9
		listindices9(t, prf);
#elif WK==5
		listindices5(t, prf);
#else
#error not implemented
#endif
		if (probdupe(prf))
			return;
		u32 soli = atomicAdd(&nsols, 1);
		if (soli < MAXSOLS)
#if WK==9
			listindices9(t, sols[soli]);
#elif WK==5
			listindices5(t, sols[soli]);
#else
#error not implemented
#endif
	}
	void showbsizes(u32 r) {
#if defined(HIST) || defined(SPARK) || defined(LOGSPARK)
		u32 ns[NBUCKETS];
		checkCudaErrors(hipMemcpy(ns, nslots[r & 1], NBUCKETS * sizeof(u32), hipMemcpyDeviceToHost));
		u32 binsizes[65];
		memset(binsizes, 0, 65 * sizeof(u32));
		for (u32 bucketid = 0; bucketid < NBUCKETS; bucketid++) {
			u32 bsize = min(ns[bucketid], NSLOTS) >> (SLOTBITS - 6);
			binsizes[bsize]++;
		}
		for (u32 i = 0; i < 65; i++) {
#ifdef HIST
			printf(" %d:%d", i, binsizes[i]);
#else
#ifdef SPARK
			u32 sparks = binsizes[i] / SPARKSCALE;
#else
			u32 sparks = 0;
			for (u32 bs = binsizes[i]; bs; bs >>= 1) sparks++;
			sparks = sparks * 7 / SPARKSCALE;
#endif
			printf("\342\226%c", '\201' + sparks);
#endif
		}
		printf("\n");
#endif
		}
	// proper dupe test is a little costly on GPU, so allow false negatives
	__device__ bool probdupe(u32 *prf) {
		unsigned short susp[PROOFSIZE];
		memset(susp, 0xffff, PROOFSIZE * sizeof(unsigned short));
		for (u32 i=0; i<PROOFSIZE; i++) {
			u32 bin = prf[i] & (PROOFSIZE-1);
			unsigned short msb = prf[i]>>WK;
			if (msb == susp[bin])
				return true;
			susp[bin] = msb;
		}
		return false;
	}
	struct htlayout {
		htalloc hta;
		u32 prevhashunits;
		u32 nexthashunits;
		u32 dunits;
		u32 prevbo;
		u32 nextbo;

		__device__ htlayout(equi *eq, u32 r) : hta(eq->hta), prevhashunits(0), dunits(0) {
			u32 nexthashbytes = hashsize(r);
			nexthashunits = hashwords(nexthashbytes);
			prevbo = 0;
			nextbo = nexthashunits * sizeof(hashunit) - nexthashbytes; // 0-3
			if (r) {
				u32 prevhashbytes = hashsize(r-1);
				prevhashunits = hashwords(prevhashbytes);
				prevbo = prevhashunits * sizeof(hashunit) - prevhashbytes; // 0-3
				dunits = prevhashunits - nexthashunits;
			}
		}
		__device__ u32 getxhash0(const slot0* pslot) const {
#ifdef XINTREE
			return pslot->attr.xhash();
#elif WN == 200 && RESTBITS == 4
			return pslot->hash->bytes[prevbo] >> 4;
#elif WN == 200 && RESTBITS == 8
			return (pslot->hash->bytes[prevbo] & 0xf) << 4 | pslot->hash->bytes[prevbo + 1] >> 4;
#elif WN == 144 && RESTBITS == 4
			return pslot->hash->bytes[prevbo] & 0xf;
#elif WN == 200 && RESTBITS == 6
			return (pslot->hash->bytes[prevbo] & 0x3) << 4 | pslot->hash->bytes[prevbo+1] >> 4;
#else
#error non implemented
#endif
		}
		__device__ u32 getxhash1(const slot1* pslot) const {
#ifdef XINTREE
			return pslot->attr.xhash();
#elif WN == 200 && RESTBITS == 4
			return pslot->hash->bytes[prevbo] & 0xf;
#elif WN == 200 && RESTBITS == 8
			return pslot->hash->bytes[prevbo];
#elif WN == 144 && RESTBITS == 4
			return pslot->hash->bytes[prevbo] & 0xf;
#elif WN == 200 && RESTBITS == 6
			return pslot->hash->bytes[prevbo] & 0x3f;
#else
#error non implemented
#endif
		}
		__device__ bool equal(const hashunit *hash0, const hashunit *hash1) const {
			return hash0[prevhashunits - 1].word == hash1[prevhashunits - 1].word;
		}
	};

	struct collisiondata {
#ifdef XBITMAP
#if NSLOTS > 64
#error cant use XBITMAP with more than 64 slots
#endif
		u64 xhashmap[NRESTS];
		u64 xmap;
#else
#if RESTBITS <= 6
		typedef uchar xslot;
#else
		typedef u16 xslot;
#endif
		static const xslot xnil = ~0;
		xslot xhashslots[NRESTS];
		xslot nextxhashslot[NSLOTS];
		xslot nextslot;
#endif
		u32 s0;

		__device__ void clear() {
#ifdef XBITMAP
			memset(xhashmap, 0, NRESTS * sizeof(u64));
#else
			memset(xhashslots, xnil, NRESTS * sizeof(xslot));
			memset(nextxhashslot, xnil, NSLOTS * sizeof(xslot));
#endif
		}
		__device__ bool addslot(u32 s1, u32 xh) {
#ifdef XBITMAP
			xmap = xhashmap[xh];
			xhashmap[xh] |= (u64)1 << s1;
			s0 = ~0;
			return true;
#else
			nextslot = xhashslots[xh];
			nextxhashslot[s1] = nextslot;
			xhashslots[xh] = s1;
			return true;
#endif
		}
		__device__ bool nextcollision() const {
#ifdef XBITMAP
			return xmap != 0;
#else
			return nextslot != xnil;
#endif
		}
		__device__ u32 slot() {
#ifdef XBITMAP
			const u32 ffs = __ffsll(xmap);
			s0 += ffs; xmap >>= ffs;
#else
			nextslot = nextxhashslot[s0 = nextslot];
#endif
			return s0;
		}
	};
		};

__global__ void digitH(equi *eq) {
	uchar hash[HASHOUT];
	blake2b_state state;
	equi::htlayout htl(eq, 0);
	const u32 hashbytes = hashsize(0); // always 23 ?
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 block = id; block < NBLOCKS; block += eq->nthreads) {
		state = eq->blake_ctx;
		blake2b_gpu_hash(&state, block, hash, HASHOUT);
		for (u32 i = 0; i<HASHESPERBLAKE; i++) {
			const uchar *ph = hash + i * WN / 8;
#if BUCKBITS == 16 && RESTBITS == 4
			const u32 bucketid = ((u32)ph[0] << 8) | ph[1];
#ifdef XINTREE
			const u32 xhash = ph[2] >> 4;
#endif
#elif BUCKBITS == 14 && RESTBITS == 6
			const u32 bucketid = ((u32)ph[0] << 6) | ph[1] >> 2;
#elif BUCKBITS == 12 && RESTBITS == 8
			const u32 bucketid = ((u32)ph[0] << 4) | ph[1] >> 4;
#elif BUCKBITS == 20 && RESTBITS == 4
			const u32 bucketid = ((((u32)ph[0] << 8) | ph[1]) << 4) | ph[2] >> 4;
#ifdef XINTREE
			const u32 xhash = ph[2] & 0xf;
#endif
#elif BUCKBITS == 12 && RESTBITS == 4
			const u32 bucketid = ((u32)ph[0] << 4) | ph[1] >> 4;
			const u32 xhash = ph[1] & 0xf;
#else
#error not implemented
#endif
			const u32 slot = atomicAdd(&eq->nslots[0][bucketid], 1);
			if (slot >= NSLOTS)
				continue;
			slot0 &s = eq->hta.trees0[0][bucketid][slot];
#ifdef XINTREE
			s.attr = tree(block*HASHESPERBLAKE+i, xhash);
#else
			s.attr = tree(block*HASHESPERBLAKE+i);
#endif
			memcpy(s.hash->bytes+htl.nextbo, ph+WN/8-hashbytes, hashbytes);
		}
	}
}

__global__ void digitO(equi *eq, const u32 r) {
	equi::htlayout htl(eq, r);
	equi::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[(r - 1) / 2][bucketid];
		u32 bsize = eq->getnslots0(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash0(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				u32 xorbucketid;
				u32 xhash;
				const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;
#if WN == 200 && BUCKBITS == 16 && RESTBITS == 4 && defined(XINTREE)
				xorbucketid = ((((u32)(bytes0[htl.prevbo] ^ bytes1[htl.prevbo]) & 0xf) << 8)
					| (bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1])) << 4
					| (xhash = bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 4;
				xhash &= 0xf;
#elif WN == 144 && BUCKBITS == 20 && RESTBITS == 4
				xorbucketid = ((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 8)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])) << 4)
					| (xhash = bytes0[htl.prevbo + 3] ^ bytes1[htl.prevbo + 3]) >> 4;
				xhash &= 0xf;
#elif WN == 96 && BUCKBITS == 12 && RESTBITS == 4
				xorbucketid = ((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 4)
					| (xhash = bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 4;
				xhash &= 0xf;
#elif WN == 200 && BUCKBITS == 14 && RESTBITS == 6
				xorbucketid = ((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) & 0xf) << 8)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])) << 2
					| (bytes0[htl.prevbo + 3] ^ bytes1[htl.prevbo + 3]) >> 6;
#else
#error not implemented
#endif
				const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot1 &xs = htl.hta.trees1[r/2][xorbucketid][xorslot];
#ifdef XINTREE
				xs.attr = tree(bucketid, s0, s1, xhash);
#else
				xs.attr = tree(bucketid, s0, s1);
#endif
				for (u32 i=htl.dunits; i < htl.prevhashunits; i++)
					xs.hash[i - htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
			}
		}
	}
}

__global__ void digitE(equi *eq, const u32 r) {
	equi::htlayout htl(eq, r);
	equi::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot1 *buck = htl.hta.trees1[(r - 1) / 2][bucketid];
		u32 bsize = eq->getnslots1(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot1 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash1(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot1 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				u32 xorbucketid;
				u32 xhash;
				const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;
#if WN == 200 && BUCKBITS == 16 && RESTBITS == 4 && defined(XINTREE)
				xorbucketid = ((u32)(bytes0[htl.prevbo] ^ bytes1[htl.prevbo]) << 8)
					| (bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]);
				xhash = (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 4;
#elif WN == 144 && BUCKBITS == 20 && RESTBITS == 4
				xorbucketid = ((((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 8)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2])) << 4)
					| (bytes0[htl.prevbo + 3] ^ bytes1[htl.prevbo + 3]) >> 4;
#elif WN == 96 && BUCKBITS == 12 && RESTBITS == 4
				xorbucketid = ((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 4)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 4;
#elif WN == 200 && BUCKBITS == 14 && RESTBITS == 6
				xorbucketid = ((u32)(bytes0[htl.prevbo + 1] ^ bytes1[htl.prevbo + 1]) << 6)
					| (bytes0[htl.prevbo + 2] ^ bytes1[htl.prevbo + 2]) >> 2;
#else
#error not implemented
#endif
				const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot0 &xs = htl.hta.trees0[r / 2][xorbucketid][xorslot];
#ifdef XINTREE
				xs.attr = tree(bucketid, s0, s1, xhash);
#else
				xs.attr = tree(bucketid, s0, s1);
#endif
				for (u32 i = htl.dunits; i < htl.prevhashunits; i++)
					xs.hash[i - htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
			}
		}
	}
}

#ifdef UNROLL
__global__ void digit_1(equi *eq) {
	equi::htlayout htl(eq, 1);
	equi::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[0][bucketid];
		u32 bsize = eq->getnslots0(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash0(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 bexor = __byte_perm(xor0, 0, 0x0123);
				const u32 xorbucketid = bexor >> 4 & BUCKMASK;
				const u32 xhash = bexor & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot1 &xs = htl.hta.trees1[0][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
				xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
				xs.hash[2].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
				xs.hash[3].word = pslot0->hash[4].word ^ pslot1->hash[4].word;
				xs.hash[4].word = pslot0->hash[5].word ^ pslot1->hash[5].word;
			}
		}
	}
}
__global__ void digit2(equi *eq) {
	equi::htlayout htl(eq, 2);
	equi::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot1 *buck = htl.hta.trees1[0][bucketid];
		u32 bsize = eq->getnslots1(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot1 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash1(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot1 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 bexor = __byte_perm(xor0, 0, 0x0123);
				const u32 xorbucketid = bexor >> 16;
				const u32 xhash = bexor >> 12 & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot0 &xs = htl.hta.trees0[1][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor0;
				xs.hash[1].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
				xs.hash[2].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
				xs.hash[3].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
				xs.hash[4].word = pslot0->hash[4].word ^ pslot1->hash[4].word;
			}
		}
	}
}
__global__ void digit3(equi *eq) {
	equi::htlayout htl(eq, 3);
	equi::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[1][bucketid];
		u32 bsize = eq->getnslots0(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash0(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
				const u32 bexor = __byte_perm(xor0, xor1, 0x1234);
				const u32 xorbucketid = bexor >> 4 & BUCKMASK;
				const u32 xhash = bexor & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot1 &xs = htl.hta.trees1[1][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor1;
				xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
				xs.hash[2].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
				xs.hash[3].word = pslot0->hash[4].word ^ pslot1->hash[4].word;
			}
		}
	}
}
__global__ void digit4(equi *eq) {
	equi::htlayout htl(eq, 4);
	equi::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot1 *buck = htl.hta.trees1[1][bucketid];
		u32 bsize = eq->getnslots1(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot1 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash1(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot1 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 bexor = __byte_perm(xor0, 0, 0x4123);
				const u32 xorbucketid = bexor >> 8;
				const u32 xhash = bexor >> 4 & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot0 &xs = htl.hta.trees0[2][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor0;
				xs.hash[1].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
				xs.hash[2].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
				xs.hash[3].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
			}
		}
	}
}
__global__ void digit5(equi *eq) {
	equi::htlayout htl(eq, 5);
	equi::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[2][bucketid];
		u32 bsize = eq->getnslots0(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash0(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
				const u32 bexor = __byte_perm(xor0, xor1, 0x2345);
				const u32 xorbucketid = bexor >> 4 & BUCKMASK;
				const u32 xhash = bexor & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot1 &xs = htl.hta.trees1[2][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor1;
				xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
				xs.hash[2].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
			}
		}
	}
}
__global__ void digit6(equi *eq) {
	equi::htlayout htl(eq, 6);
	equi::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot1 *buck = htl.hta.trees1[2][bucketid];
		u32 bsize = eq->getnslots1(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot1 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash1(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot1 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
				const u32 bexor = __byte_perm(xor0, xor1, 0x2345);
				const u32 xorbucketid = bexor >> 16;
				const u32 xhash = bexor >> 12 & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot0 &xs = htl.hta.trees0[3][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor1;
				xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
			}
		}
	}
}
__global__ void digit7(equi *eq) {
	equi::htlayout htl(eq, 7);
	equi::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[3][bucketid];
		u32 bsize = eq->getnslots0(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash0(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 bexor = __byte_perm(xor0, 0, 0x4012);
				const u32 xorbucketid = bexor >> 4 & BUCKMASK;
				const u32 xhash = bexor & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot1 &xs = htl.hta.trees1[3][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor0;
				xs.hash[1].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
			}
		}
	}
}
__global__ void digit8(equi *eq) {
	equi::htlayout htl(eq, 8);
	equi::collisiondata cd;
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot1 *buck = htl.hta.trees1[3][bucketid];
		u32 bsize = eq->getnslots1(bucketid);
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot1 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash1(pslot1)))
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot1 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash))
					continue;
				const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
				const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
				const u32 bexor = __byte_perm(xor0, xor1, 0x3456);
				const u32 xorbucketid = bexor >> 16;
				const u32 xhash = bexor >> 12 & 0xf;
				const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
				if (xorslot >= NSLOTS)
					continue;
				slot0 &xs = htl.hta.trees0[4][xorbucketid][xorslot];
				xs.attr = tree(bucketid, s0, s1, xhash);
				xs.hash[0].word = xor1;
			}
		}
	}
}
#endif

__global__ void digitK(equi *eq) {
	equi::collisiondata cd;
	equi::htlayout htl(eq, WK);
	const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
	for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
		cd.clear();
		slot0 *buck = htl.hta.trees0[(WK - 1) / 2][bucketid];
		u32 bsize = eq->getnslots0(bucketid); // assume WK odd
		for (u32 s1 = 0; s1 < bsize; s1++) {
			const slot0 *pslot1 = buck + s1;
			if (!cd.addslot(s1, htl.getxhash0(pslot1))) // assume WK odd
				continue;
			for (; cd.nextcollision();) {
				const u32 s0 = cd.slot();
				const slot0 *pslot0 = buck + s0;
				if (htl.equal(pslot0->hash, pslot1->hash)) {
#ifdef XINTREE
					eq->candidate(tree(bucketid, s0, s1, 0));
#else
					eq->candidate(tree(bucketid, s0, s1));
#endif
				}
			}
		}
	}
}




eq_cuda_context::eq_cuda_context(int tpb, int blocks, int id)
	: threadsperblock(tpb), totalblocks(blocks), device_id(id)
{
	eq = new equi(threadsperblock * totalblocks);
	sol_memory = malloc(sizeof(proof) * MAXSOLS + 4096);
	solutions = (proof*)(((long long)sol_memory + 4095) & -4096);

	checkCudaErrors(hipSetDevice(device_id));
	checkCudaErrors(hipDeviceReset());
	checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
	checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	checkCudaErrors(hipMalloc((void**)&heap0, sizeof(digit0)));
	checkCudaErrors(hipMalloc((void**)&heap1, sizeof(digit1)));
	for (u32 r = 0; r < WK; r++)
		if ((r & 1) == 0)
			eq->hta.trees0[r / 2] = (bucket0 *)(heap0 + r / 2);
		else
			eq->hta.trees1[r / 2] = (bucket1 *)(heap1 + r / 2);

	checkCudaErrors(hipMalloc((void**)&eq->nslots, 2 * NBUCKETS * sizeof(u32)));
	checkCudaErrors(hipMalloc((void**)&eq->sols, MAXSOLS * sizeof(proof)));

	checkCudaErrors(hipMalloc((void**)&device_eq, sizeof(equi)));
}


eq_cuda_context::~eq_cuda_context()
{
	/*checkCudaErrors(hipFree(eq->nslots));
	checkCudaErrors(hipFree(eq->sols));
	checkCudaErrors(hipFree(eq->hta.trees0[0]));
	checkCudaErrors(hipFree(eq->hta.trees1[0]));*/
	checkCudaErrors(hipSetDevice(device_id));
	checkCudaErrors(hipDeviceReset());
	free(sol_memory);
	delete eq;
}


void eq_cuda_context::solve(const char *tequihash_header,
	unsigned int tequihash_header_len,
	const char* nonce,
	unsigned int nonce_len,
	std::function<bool()> cancelf,
	std::function<void(const std::vector<uint32_t>&, size_t, const unsigned char*)> solutionf,
	std::function<void(void)> hashdonef)
{
	checkCudaErrors(hipSetDevice(device_id));

	eq->setheadernonce(tequihash_header, tequihash_header_len, nonce, nonce_len);
	checkCudaErrors(hipMemcpy(device_eq, eq, sizeof(equi), hipMemcpyHostToDevice));

	digitH << <totalblocks, threadsperblock >> >(device_eq);
	if (cancelf()) return;
#if BUCKBITS == 16 && RESTBITS == 4 && defined XINTREE && defined(UNROLL)
	digit_1 << <totalblocks, threadsperblock >> >(device_eq);
	if (cancelf()) return;
	digit2 << <totalblocks, threadsperblock >> >(device_eq);
	if (cancelf()) return;
	digit3 << <totalblocks, threadsperblock >> >(device_eq);
	if (cancelf()) return;
	digit4 << <totalblocks, threadsperblock >> >(device_eq);
	if (cancelf()) return;
	digit5 << <totalblocks, threadsperblock >> >(device_eq);
	if (cancelf()) return;
	digit6 << <totalblocks, threadsperblock >> >(device_eq);
	if (cancelf()) return;
	digit7 << <totalblocks, threadsperblock >> >(device_eq);
	if (cancelf()) return;
	digit8 << <totalblocks, threadsperblock >> >(device_eq);
#else
	for (u32 r = 1; r < WK; r++) {
		r & 1 ? digitO << <totalblocks, threadsperblock >> >(device_eq, r)
			: digitE << <totalblocks, threadsperblock >> >(device_eq, r);
	}
#endif
	if (cancelf()) return;
	digitK << <totalblocks, threadsperblock >> >(device_eq);

	checkCudaErrors(hipMemcpy(eq, device_eq, sizeof(equi), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(solutions, eq->sols, MAXSOLS * sizeof(proof), hipMemcpyDeviceToHost));

	for (unsigned s = 0; (s < eq->nsols) && (s < MAXSOLS); s++)
	{
		std::vector<uint32_t> index_vector(PROOFSIZE);
		for (u32 i = 0; i < PROOFSIZE; i++) {
			index_vector[i] = solutions[s][i];
		}

		solutionf(index_vector, DIGITBITS, nullptr);
		if (cancelf()) return;
	}
	hashdonef();
}